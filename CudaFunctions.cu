#include "hip/hip_runtime.h"
/*!
 * \file CudaFunctions.cu
 * \author Libor Bukata
 * \brief RCPSP Cuda functions.
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "CudaConstants.h"
#include "CudaFunctions.cuh"

#if defined _WIN32 || defined _WIN64 || defined WIN32 || defined WIN64
#include "hip/hip_runtime.h"
#include ""
#endif

using std::cerr;
using std::cout;
using std::endl;

//! Texture reference of activities resource requirements.
texture<uint8_t,1,hipReadModeElementType> cudaActivitiesResourcesTex;
//! Texture reference of predecessors.
texture<uint16_t,1,hipReadModeElementType> cudaPredecessorsTex;
//! Texture reference of predecessors indices.
texture<uint16_t,1,hipReadModeElementType> cudaPredecessorsIndicesTex;
//! Texture reference of successors.
texture<uint16_t,1,hipReadModeElementType> cudaSuccessorsTex;
//! Texture reference of successors indices.
texture<uint16_t,1,hipReadModeElementType> cudaSuccessorsIndicesTex;

//! The longest paths from the end dummy activity to the others in the transformed graph.
__constant__ uint16_t rightLeftLongestPaths[NUMBER_OF_ACTIVITIES];


/* CUDA BIND TEXTURES */

int bindTexture(void *texData, int32_t arrayLength, int option)	{
	switch (option)	{
		case ACTIVITIES_RESOURCES:
			return hipBindTexture(NULL, cudaActivitiesResourcesTex, texData, arrayLength*sizeof(uint8_t));
		case PREDECESSORS:
			return hipBindTexture(NULL, cudaPredecessorsTex, texData, arrayLength*sizeof(uint16_t));
		case PREDECESSORS_INDICES:
			return hipBindTexture(NULL, cudaPredecessorsIndicesTex, texData, arrayLength*sizeof(uint16_t));
		case SUCCESSORS:
			return hipBindTexture(NULL, cudaSuccessorsTex, texData, arrayLength*sizeof(uint16_t));
		case SUCCESSORS_INDICES:
			return hipBindTexture(NULL, cudaSuccessorsIndicesTex, texData, arrayLength*sizeof(uint16_t));
		default:
			cerr<<"bindTextures: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}

int unbindTexture(int option)	{
	switch (option)	{
		case ACTIVITIES_RESOURCES:
			return hipUnbindTexture(cudaActivitiesResourcesTex);
		case PREDECESSORS:
			return hipUnbindTexture(cudaPredecessorsTex);
		case PREDECESSORS_INDICES:
			return hipUnbindTexture(cudaPredecessorsIndicesTex);
		case SUCCESSORS:
			return hipUnbindTexture(cudaSuccessorsTex);
		case SUCCESSORS_INDICES:
			return hipUnbindTexture(cudaSuccessorsIndicesTex);
		default:
			cerr<<"unbindTextures: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}

int memcpyToSymbol(void *source, int32_t arrayLength, int option)	{
	switch (option)	{
		case THE_LONGEST_PATHS:
			return hipMemcpyToSymbol(HIP_SYMBOL(rightLeftLongestPaths), (void*) source, arrayLength*sizeof(uint16_t));
		default:
			cerr<<"memcpyToSymbol: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}


/* CUDA IMPLEMENT OF SOURCES LOAD - CAPACITY RESOLUTION  */

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \brief Prepare arrays for next use (schedule evaluation).
 */
inline __device__ void cudaPrepareArrays(const CudaData& cudaData, uint16_t *& resourcesLoad, uint16_t *& startValues)	{
	for (uint16_t i = 0; i < cudaData.sumOfCapacities; ++i)
		resourcesLoad[i] = 0;
	for (uint16_t i = 0; i < cudaData.maximalCapacityOfResource; ++i)
		startValues[i] = 0;
}

/*!
 * \param numberOfResources Number of resources.
 * \param activityId Activity identification.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param resourceIndices Access indices for resources.
 * \return Earliest start time of an activity.
 * \brief Function return earliest possible start time of an activity. Precedence relations are ignored.
 */
inline __device__ uint16_t cudaGetEarliestStartTime(const uint16_t& numberOfResources, const uint16_t& activityId, uint16_t *&resourcesLoad, uint16_t *&resourceIndices) {
	uint16_t bestStart = 0;
	for (uint8_t resourceId = 0; resourceId < numberOfResources; ++resourceId)	{
		uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		if (activityRequirement > 0)
			bestStart = max(resourcesLoad[resourceIndices[resourceId+1]-activityRequirement], bestStart);
	}
	return bestStart;
}

/*!
 * \param activityId Activity identification.
 * \param activityStart Start time of an activity.
 * \param activityStop Stop time of an activity.
 * \param numberOfResources Number of resources.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \brief Function add new activity and update resources arrays. Irreversible process.
 */
inline __device__ void cudaAddActivity(const uint16_t& activityId, const uint16_t& activityStart, const uint16_t& activityStop,
		const uint16_t& numberOfResources, uint16_t *&resourceIndices,  uint16_t *&resourcesLoad, uint16_t *&startValues)	{
	
	int32_t requiredSquares, timeDiff;
	int32_t c, k, capacityOfResource, resourceRequirement, newStartTime, resourceStartIdx;
	for (uint8_t resourceId = 0; resourceId < numberOfResources; ++resourceId)	{
		resourceStartIdx = resourceIndices[resourceId];
		capacityOfResource = resourceIndices[resourceId+1]-resourceStartIdx;
		resourceRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		requiredSquares = resourceRequirement*(activityStop-activityStart);
		if (requiredSquares > 0)	{
			c = 0; k = 0;
			newStartTime = activityStop;
			while (requiredSquares > 0 && k < capacityOfResource)	{
				if (resourcesLoad[resourceStartIdx+k] < newStartTime)	{
					if (c >= resourceRequirement)
						newStartTime = startValues[c-resourceRequirement];
					timeDiff = newStartTime-max(resourcesLoad[resourceStartIdx+k], activityStart);
					if (requiredSquares-timeDiff > 0)	{
						requiredSquares -= timeDiff;
						startValues[c++] = resourcesLoad[resourceStartIdx+k];
						resourcesLoad[resourceStartIdx+k] = newStartTime;
					} else {
						resourcesLoad[resourceStartIdx+k] = newStartTime-timeDiff+requiredSquares;
						break;
					}
				}
				++k;
			}
		}
	}
}

/* CUDA IMPLEMENT OF SOURCES LOAD - TIME RESOLUTION  */

/*!
 * \param numberOfActivities Number of activities in the project.
 * \param numberOfResources Number of renewable resources in the project.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param resourceIndices Access indices for resources.
 * \brief It initializes vectors of free capacities to initial values (capacities of resources).
 */
inline __device__ void cudaPrepareArrays(const uint16_t& numberOfActivities, const uint16_t& numberOfResources, const uint32_t& UBTime,
	       	uint8_t *& remainingResourcesCapacity, uint16_t *& resourceIndices)	{
	for (uint16_t resourceId = 0; resourceId < numberOfResources; ++resourceId)
		for (uint32_t t = 0; t < UBTime; ++t)
			remainingResourcesCapacity[resourceId*UBTime+t] = resourceIndices[resourceId+1]-resourceIndices[resourceId];
}

/*!
 * \param numberOfResources Number of renewable resources in the project.
 * \param activityId Identification of the activity that should be added (required for texture memory access).
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param precTime The earliest precedence violation free start time of the activity activityId.
 * \param activityDuration Duration of the activity activityId.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \return The earliest start time of the activity without resource overload.
 * \brief It finds out the earliest start time of the activity activityId.
 */
inline __device__ uint16_t cudaGetEarliestStartTime(const uint16_t& numberOfResources, const uint16_t& activityId,
		uint8_t *&remainingResourcesCapacity, const uint16_t& precTime, int32_t activityDuration, const uint32_t& UBTime) {
	int32_t loadTime = 0, t = UBTime;
	for (t = precTime; t < UBTime && loadTime < activityDuration; ++t)       {
		bool capacityAvailable = true;
		for (int32_t resourceId = 0; resourceId < numberOfResources && capacityAvailable; ++resourceId)        {
			uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
			if (remainingResourcesCapacity[resourceId*UBTime+t] < activityRequirement)	{
				loadTime = 0;
				capacityAvailable = false;
			}
		}
		if (capacityAvailable == true)
			++loadTime;
	}
	return (uint16_t) t-loadTime;
}

/*!
 * \param activityId Identification of the added activity.
 * \param activityStart Scheduled start time of the activity.
 * \param activityStop Scheduled finish time of the activity.
 * \param numberOfResources Number of renewable resources in the project.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \brief It updates the state of all resources after activity is added.
 */
inline __device__ void cudaAddActivity(const uint16_t& activityId, const uint16_t& activityStart, const uint16_t& activityStop,
		const uint16_t& numberOfResources, uint8_t *&remainingResourcesCapacity, const uint32_t& UBTime)	{
	for (int32_t resourceId = 0; resourceId < numberOfResources; ++resourceId)     {
		uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		for (uint32_t t = activityStart; t < activityStop; ++t)
			remainingResourcesCapacity[resourceId*UBTime+t] -= activityRequirement;
	}
}

/* CUDA IMPLEMENTATION OF THE BASE RESOURCE EVALUATION FUNCTIONS */

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param blockOrder Current order of the activities.
 * \param indexI Swap index i.
 * \param indexJ Swap index j.
 * \param activitiesDuration Duration of the activities.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param startTimesWriterById Array of start times of the scheduled activities ordered by ID's.
 * \param capacityResolution If true then capacity based algorithm is selected else time based algorithm is selected.
 * \param forward It determines if schedule is forward or backward evaluated.
 * \return Schedule length without any penalties.
 * \brief Function evaluate schedule and return total schedule length.
 */
__device__ uint16_t cudaEvaluateOrder(const CudaData& cudaData, uint16_t *&blockOrder, const uint16_t& indexI, const uint16_t& indexJ, uint8_t *&activitiesDuration, uint16_t *&resourceIndices,
		uint16_t *resourcesLoad, uint16_t *startValues, uint8_t *remainingResourcesCapacity, uint16_t *startTimesWriterById, bool capacityResolution, bool forward = true)	{

	// Current cost of the schedule.
	uint16_t scheduleLength = 0;

	// Init state of resources.
	if (capacityResolution == true)
		cudaPrepareArrays(cudaData, resourcesLoad, startValues);
	else
		cudaPrepareArrays(cudaData.numberOfActivities, cudaData.numberOfResources, MAXIMAL_SUM_OF_FLOATS, remainingResourcesCapacity, resourceIndices);
	
	for (uint16_t i = 0; i < cudaData.numberOfActivities; ++i)	{

		uint16_t index = ((forward == true) ? i : cudaData.numberOfActivities-i-1);
		uint16_t activityId = blockOrder[index];

		// Logical swap.
		if (index == indexI)
			activityId = blockOrder[indexJ];

		if (index == indexJ)
			activityId = blockOrder[indexI];

		// Get the earliest start time without precedence penalty. (if moves are precedence penalty free)
		uint16_t start = 0;
		uint16_t baseIndex;
		uint16_t numberOfRelatedActivities;
		if (forward == true) {
			baseIndex = tex1Dfetch(cudaPredecessorsIndicesTex, activityId);
			numberOfRelatedActivities = tex1Dfetch(cudaPredecessorsIndicesTex, activityId+1)-baseIndex;
		} else	{
			baseIndex = tex1Dfetch(cudaSuccessorsIndicesTex, activityId);
			numberOfRelatedActivities = tex1Dfetch(cudaSuccessorsIndicesTex, activityId+1)-baseIndex;
		}
		for (uint16_t j = 0; j < numberOfRelatedActivities; ++j)	{
			uint16_t relatedActivityId;
			if (forward == true)
				relatedActivityId = tex1Dfetch(cudaPredecessorsTex, baseIndex+j);
			else
				relatedActivityId = tex1Dfetch(cudaSuccessorsTex, baseIndex+j);
			start = max(startTimesWriterById[relatedActivityId]+activitiesDuration[relatedActivityId], start);
		}

		// Get the earliest start time if the resources restrictions are counted.
		if (capacityResolution == true)
			start = max(cudaGetEarliestStartTime(cudaData.numberOfResources, activityId, resourcesLoad, resourceIndices), start);
		else
			start = max(cudaGetEarliestStartTime(cudaData.numberOfResources, activityId, remainingResourcesCapacity,
						start, activitiesDuration[activityId], MAXIMAL_SUM_OF_FLOATS), start);

		// Add activity = update resources arrays + write start time.
		uint16_t stop = start+activitiesDuration[activityId];
		if (capacityResolution == true)
			cudaAddActivity(activityId, start, stop, cudaData.numberOfResources, resourceIndices, resourcesLoad, startValues);
		else
			cudaAddActivity(activityId, start, stop, cudaData.numberOfResources, remainingResourcesCapacity, MAXIMAL_SUM_OF_FLOATS);
		scheduleLength = max(scheduleLength, stop);

		startTimesWriterById[activityId] = start;
	}

	return scheduleLength;
}

/*!
 * \param order Order of activities.
 * \param timeValuesById Time values of activities. Accessed through the identifications of activities.
 * \param size Length of the order and timeValuesById arrays.
 * \brief It reorders input order in accordance with timeValuesById array. It's stable sort with algorithm complexity O(n^2).
 */
inline __device__ void cudaInsertSort(uint16_t* order, const uint16_t * const& timeValuesById, const int16_t& size)	{
	for (int32_t i = 1; i < size; ++i)	{
		for (int32_t j = i; (j > 0) && ((timeValuesById[order[j]] < timeValuesById[order[j-1]]) == true); --j)	{
			uint16_t t = order[j];
			order[j] = order[j-1];
			order[j-1] = t;
		}
	}
}

/*!
 * \param order Order of activities.
 * \param startTimesById Start time values of activities. Accessed through the identifications of activities.
 * \param size Length of the order and timeValuesById arrays.
 * \brief It converts startTimesById array to activities order.
 */
inline __device__ void cudaConvertStartTimesById2ActivitiesOrder(uint16_t *& order, uint16_t *startTimesById, uint16_t size)	{
	cudaInsertSort(order, startTimesById, size);
}

/* CHECK PRECEDENCE FUNCTIONS */

/*!
 * \param successorsMatrix Bit matrix of successors.
 * \param numberOfActivities Number of activities.
 * \param activityId1 Activity identification.
 * \param activityId2 Activity identification.
 * \return True if an activity with identification activityId2 is successor of an activity with identification activityId1.
 * \brief Check if activity ID2 is successor of activity ID1.
 */
inline __device__ bool cudaGetMatrixBit(const uint8_t * const& successorsMatrix, const uint16_t& numberOfActivities, const int16_t& activityId1, const int16_t& activityId2)	{
	uint32_t bitPossition = activityId1*numberOfActivities+activityId2;
	if ((successorsMatrix[bitPossition/8] & (1<<(bitPossition % 8))) > 0)
		return true;
	else
		return false;
}

/*!
 * \param order Sequence of activities.
 * \param successorsMatrix Bit matrix of successors.
 * \param numberOfActivities Number of activities.
 * \param i Index i of swap.
 * \param j Index j of swap.
 * \param light If true then light version is executed. (precedences from activity at index i aren't checked)
 * \return True if current swap won't break relation precedences else false.
 * \brief Check if requested move is precedence penalty free.
 */
__device__ bool cudaCheckSwapPrecedencePenalty(const uint16_t * const& order, const uint8_t * const& successorsMatrix, const uint16_t& numberOfActivities, int16_t i, int16_t j, bool light = false)	{
	if (i > j)	{
		int16_t t = i;
		i = j; j = t;
	}
	for (uint16_t k = i; k < j; ++k)	{
		if (cudaGetMatrixBit(successorsMatrix, numberOfActivities, order[k], order[j]) == true)
			return false;
	}
	if (!light)	{
		for (uint16_t k = i+1; k <= j; ++k)	{
			if (cudaGetMatrixBit(successorsMatrix, numberOfActivities, order[i], order[k]) == true)
				return false;
		}
	}
	return true;
}

/*!
 * \param numAct The number of activities.
 * \param successorsMatrix Binary matrix of successors.
 * \param activitiesDuration Duration of each activity.
 * \param startTimesById Array of start time values of the scheduled activities ordered by ID's.
 * \return The precedence penalty.
 * \brief It finds out all precedence penalties and computes penalty.
 * \note The penalty should be zero since only non-precedence breaking moves are allowed.
 */
__device__ uint32_t cudaComputePrecedencePenalty(uint16_t numAct, uint8_t *successorsMatrix, uint8_t *activitiesDuration, uint16_t *startTimesById)  {
	uint32_t penalty = 0;
	for (uint16_t id1 = 0; id1 < numAct; ++id1)        {
		for (uint16_t id2 = 0; id2 < numAct; ++id2)        {
			if (id1 != id2 && cudaGetMatrixBit(successorsMatrix, numAct, id1, id2) == true)	{
				if (startTimesById[id1]+activitiesDuration[id1] > startTimesById[id2])
					penalty += startTimesById[id1]+activitiesDuration[id1]-startTimesById[id2];
			}

		}
	}
	return penalty;
}

/* SOFT VIOLATION PENALTIES */
#include <cstdio>
/*!
 * \param numberOfActivities The number of the activities in the project.
 * \param activitiesDuration Duration of each activity.
 * \param makespan The best known project makespan.
 * \param startTimesById Array of start time values of the scheduled activities ordered by ID's.
 * \return It returns overall tardiness penalty.
 */
__device__ uint32_t cudaComputeTardinessPenalty(uint16_t numberOfActivities, uint8_t *activitiesDuration, uint32_t makespan, uint16_t *startTimesById)	{
	uint32_t overhangPenalty = 0;
	for (uint16_t id = 0; id < numberOfActivities; ++id)	{
		if (startTimesById[id]+activitiesDuration[id]+rightLeftLongestPaths[id] > makespan)
			overhangPenalty += startTimesById[id]+activitiesDuration[id]+rightLeftLongestPaths[id]-makespan;
	}
	return overhangPenalty;
}

/*!
 * \param cudaData RCPSP constants, variables and data.
 * \param addedEdges Extra edges added to each solution in the solution set.
 * \param startTimesById Array of start time values computed by the evaluation algorithm.
 * \return The precedence penalty.
 */
__device__ uint32_t cudaComputePenaltyOfEdgeViolations(const CudaData& cudaData, Edge *& addedEdges, uint16_t *startTimesById)	{
	uint32_t precedencePenalty = 0;
	for (uint32_t e = 0; e < cudaData.numberOfAddedEdges; ++e)	{
		if (startTimesById[addedEdges[e].i]+addedEdges[e].weight > startTimesById[addedEdges[e].j])
			precedencePenalty += startTimesById[addedEdges[e].i]+addedEdges[e].weight-startTimesById[addedEdges[e].j];
	}
	if (precedencePenalty > 1000)
		printf("penalty: %d\n", precedencePenalty);
	return precedencePenalty;
}

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param blockOrder Order of activities.
 * \param bestScheduleStartTimesById Start time values of activities for the best shaked schedule.
 * \param activitiesDuration Duration of each activity.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \return The makespan of the best found shaked schedule.
 * \brief Iterative method tries to shake down activities in the schedule to ensure equally loaded resources. 
 * Therefore, the shorter schedule could be found.
 */
__device__ uint16_t cudaShakingDownEvaluation(const CudaData& cudaData, uint16_t *& blockOrder, uint16_t *bestScheduleStartTimesById, uint8_t *& activitiesDuration, uint16_t *& resourceIndices,
		uint16_t *resourcesLoad, uint16_t *startValues, uint8_t *remainingResourcesCapacity)	{

	uint16_t bestScheduleLength = 0xffff;
	uint16_t *currentOrder = new uint16_t[cudaData.numberOfActivities];
	if (!currentOrder)
		return bestScheduleLength;
	uint16_t *timeValuesById = new uint16_t[cudaData.numberOfActivities];
	if (!timeValuesById)	{
		delete[] currentOrder;
		return bestScheduleLength;
	}

	for (uint16_t i = 0; i < cudaData.numberOfActivities; ++i)
		currentOrder[i] = blockOrder[i];

	while (true)	{
		int32_t scheduleLength = cudaEvaluateOrder(cudaData, currentOrder, 0xffff, 0xffff, activitiesDuration, resourceIndices,
			       	resourcesLoad, startValues, remainingResourcesCapacity, timeValuesById, false, true);

		if (scheduleLength < bestScheduleLength)	{
			bestScheduleLength = scheduleLength;
			if (bestScheduleStartTimesById != NULL)	{
				for (uint16_t id = 0; id < cudaData.numberOfActivities; ++id)
					bestScheduleStartTimesById[id] = timeValuesById[id];
			}
		} else {
			break;
		}

		for (uint16_t id = 0; id < cudaData.numberOfActivities; ++id)
			timeValuesById[id] += activitiesDuration[id];

		cudaInsertSort(currentOrder, timeValuesById, cudaData.numberOfActivities);

		int32_t scheduleLengthBackward = cudaEvaluateOrder(cudaData, currentOrder, 0xffff, 0xffff, activitiesDuration,
			       	resourceIndices, resourcesLoad, startValues, remainingResourcesCapacity, timeValuesById, false, false);
		int32_t diffCmax = scheduleLength-scheduleLengthBackward;

		for (uint32_t id = 0; id < cudaData.numberOfActivities; ++id)
			timeValuesById[id] = scheduleLengthBackward-timeValuesById[id]-activitiesDuration[id];

		for (uint32_t id = 0; id < cudaData.numberOfActivities; ++id)	{
			if (((int32_t) timeValuesById[id])+diffCmax > 0)
				timeValuesById[id] += diffCmax;
			else
				timeValuesById[id] = 0;
		}

		cudaInsertSort(currentOrder, timeValuesById, cudaData.numberOfActivities);
	}

	delete[] timeValuesById;
	delete[] currentOrder;

	return bestScheduleLength;
}


/*	CUDA IMPLEMENT OF SIMPLE TABU LIST */

/*!
 * \param numberOfActivities Number of activities.
 * \param i Swap index i.
 * \param j Swap index j.
 * \param tabuCache Block tabu cache - fast check if move is in tabu list.
 * \return True if move is possible else false.
 * \brief Check if move is in tabu list.
 */
inline __device__ bool cudaIsPossibleMove(const uint16_t& numberOfActivities, const uint16_t& i, const uint16_t& j, uint8_t *&tabuCache)	{
	if (tabuCache[i*numberOfActivities+j] == 0 || tabuCache[j*numberOfActivities+i] == 0)
		return true;
	else
		return false;
}

/*!
 * \param numberOfActivities Number of activities.
 * \param i Swap index i of added move.
 * \param j Swap index j of added move.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param tabuIdx Current index at tabu list.
 * \param tabuListSize Tabu list size.
 * \brief Add specified move to tabu list and update tabu cache.
 */
inline __device__ void cudaAddTurnToTabuList(const uint16_t& numberOfActivities, const uint16_t& i, const uint16_t& j, MoveIndices *&tabuList, uint8_t *&tabuCache, uint16_t& tabuIdx, const uint16_t& tabuListSize)	{

	MoveIndices move = tabuList[tabuIdx];
	uint16_t iOld = move.i, jOld = move.j;

	if (iOld != 0 && jOld != 0)
		tabuCache[iOld*numberOfActivities+jOld] = tabuCache[jOld*numberOfActivities+iOld] = 0;

	move.i = i; move.j = j;
	tabuList[tabuIdx] = move;
	tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 1;

	tabuIdx = (tabuIdx+1) % tabuListSize;
}


/* HELP FUNCTIONS */

/*!
 * \param numberOfActivities Number of activities.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param numberOfElements Number of tabu list elements that will be removed.
 * \brief Remove specified number of elements from tabu list and update tabu cache.
 */
inline __device__ void cudaClearTabuList(const uint16_t& numberOfActivities, MoveIndices *tabuList, uint8_t *tabuCache, const uint16_t& numberOfElements)	{
	for (uint16_t k = threadIdx.x; k < numberOfElements; k += blockDim.x)	{
		MoveIndices *tabuMove = &tabuList[k];
		uint16_t i = tabuMove->i, j = tabuMove->j;
		tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 0;
		tabuMove->j = tabuMove->i = 0;
	}
	__syncthreads();
	return;
}

/*!
 * \param numberOfActivities Number of activities.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param tabuListSize Block tabu list size.
 * \param blockOrder Block schedule - order.
 * \param externalSolution Solution from a set or the best global solution. (order)
 * \param externalTabuList Tabu list of external solution.
 * \brief Replace current block solution with a read external solution (order+tabu).
 */
inline __device__ void cudaReadExternalSolution(const uint16_t& numberOfActivities, MoveIndices *tabuList, uint8_t *tabuCache, const uint16_t& tabuListSize,
		uint16_t *blockOrder, uint16_t *externalSolution, MoveIndices *externalTabuList)	{
	// Clear current tabu list and tabu cache.
	cudaClearTabuList(numberOfActivities, tabuList, tabuCache, tabuListSize);
	// Read block order.
	for (uint16_t i = threadIdx.x; i < numberOfActivities; i += blockDim.x)
		blockOrder[i] = externalSolution[i];
	// Read block tabu list and create tabu cache.
	for (uint16_t l = threadIdx.x; l < tabuListSize; l += blockDim.x)	{
		tabuList[l] = externalTabuList[l];
		MoveIndices *move = &tabuList[l];
		uint16_t i = move->i, j = move->j;
		tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 1;
	}
	__syncthreads();
	return;
}

/* REORDER ARRAY FUNCTION */

/*!
 * \tparam T uint16_t or uint32_t.
 * \param moves Array of moves which should be reorder.
 * \param resultMerge Result array of reordered moves.
 * \param threadsCounter Helper array for threads counters.
 * \param size How many elements will be processed at moves array.
 * \return Number of written elements to resultMerge array.
 * \brief Move all valid moves to the resultMerge array and return number of valid moves.
 */
template <typename T>
inline __device__ uint32_t cudaReorderMoves(uint32_t *moves, uint32_t *resultMerge, T *threadsCounter, const uint32_t& size)	{
	threadsCounter[threadIdx.x] = 0;
	uint32_t threadAmount = size/blockDim.x+1;
	for (uint32_t i = threadIdx.x*threadAmount; i < size && i < (threadIdx.x+1)*threadAmount; ++i)	{
		if (moves[i] != 0)
			++threadsCounter[threadIdx.x];
	}
	__syncthreads();
	for (uint32_t k = 0; (1<<k) < blockDim.x; ++k)   {
		uint32_t step = 1<<k;
		uint32_t begIdx = (step-1)+2*step*threadIdx.x;
		if (begIdx < blockDim.x-step)
			threadsCounter[begIdx+step] += threadsCounter[begIdx];
		__syncthreads();
	}
	for (int32_t k = (blockDim.x>>1); k > 1; k >>= 1)	{
		uint32_t step = k/2;
		uint32_t begIdx = (k-1)+2*step*threadIdx.x;
		if (begIdx < blockDim.x-step) 
			threadsCounter[begIdx+step] += threadsCounter[begIdx];
		__syncthreads();
	}
	uint32_t threadStartIndex = threadIdx.x > 0 ? threadsCounter[threadIdx.x-1] : 0;
	for (uint32_t i = threadIdx.x*threadAmount; i < size && i < (threadIdx.x+1)*threadAmount; ++i)	{
		if (moves[i] != 0)
			resultMerge[threadStartIndex++] = moves[i];
	}
	__syncthreads();
	return threadsCounter[blockDim.x-1];
}

/* DIVERSIFICATION FUNCTION */

/*!
 * \param numberOfActivities Number of activities.
 * \param order Current schedule - sequence of activities.
 * \param successorsMatrix Bit matrix of successors.
 * \param diversificationSwaps Number of diversification swaps.
 * \param state State of the random generator.
 * \brief Function performs specified number of precedence penalty free swaps.
 */
inline __device__ void cudaDiversificationOfSolution(const uint16_t& numberOfActivities, uint16_t *order, const uint8_t *successorsMatrix, const uint32_t& diversificationSwaps, hiprandState *state)	{
		
	uint32_t performedSwaps = 0;
	while (performedSwaps < diversificationSwaps)  {
		uint16_t i = (hiprand(state) % (numberOfActivities-2)) + 1;
		uint16_t j = (hiprand(state) % (numberOfActivities-2)) + 1;
		if ((i != j) && (cudaCheckSwapPrecedencePenalty(order, successorsMatrix, numberOfActivities, i, j) == true))	{
			uint16_t t = order[i];
			order[i] = order[j];
			order[j] = t;
			++performedSwaps;
		}
	}
	return;
}


/*	CUDA IMPLEMENT OF GLOBAL KERNEL */

/*!
 * Global function for RCPSP problem. Blocks communicate with each other through global memory.
 * Local variables are coalesced. Dynamic shared memory and texture memory is used.
 * \param cudaData All required constants, pointers to device memory, setting variables, ....
 * \brief Solve RCPSP problem on GPU.
 */
__global__ void cudaSolveRCPSP(const CudaData cudaData)	{
	
	__shared__ bool initialized;
	__shared__ uint32_t iter;
	__shared__ MoveInfo iterBestMove;
	__shared__ Edge *blockAddedEdges;
	__shared__ uint32_t blockBestCost;
	__shared__ uint16_t *blockBestSolution;
	__shared__ uint32_t maximalNeighbourhoodSize;
	__shared__ uint8_t *blockActivitiesDuration;
	__shared__ uint16_t *blockCurrentOrder;
	__shared__ uint8_t *blockSuccessorsMatrix;
	__shared__ MoveInfo *blockMergeArray;
	__shared__ uint16_t *blockPartitionCounterUInt16;
	__shared__ uint32_t *blockPartitionCounterUInt32;
	__shared__ MoveIndices *blockReorderingArray;
	__shared__ MoveIndices *blockReorderingArrayHelp;

	__shared__ uint16_t blockTabuIdx;
	__shared__ uint16_t blockTabuListSize;
	__shared__ MoveIndices *blockTabuList;
	__shared__ uint8_t *blockTabuCache;
	__shared__ int32_t blockIndexOfSetSolution;
	__shared__ bool blockReadPossible;
	__shared__ bool blockWriteBestBlock;
	__shared__ bool blockReadSetSolution;
	__shared__ bool blockWriteSetSolution;
	__shared__ bool blockCriticalPathLengthAchieved;
	__shared__ uint32_t blockNumberOfIterationsSinceBest;
	__shared__ uint32_t blockMaximalNumberOfIterationsSinceBest;
	__shared__ uint16_t *blockResourceIndices;

	__shared__ hiprandState randState;

	hiprandState threadRandState;
	hiprand_init(blockDim.x*blockIdx.x+threadIdx.x, threadIdx.x, 0, &threadRandState);

	uint16_t threadResourcesLoad[TOTAL_SUM_OF_CAPACITY];
	uint16_t threadStartValues[MAXIMUM_CAPACITY_OF_RESOURCE];
	uint8_t threadRemainingResourcesCapacity[NUMBER_OF_RESOURCES*MAXIMAL_SUM_OF_FLOATS];
	uint16_t threadStartTimesById[NUMBER_OF_ACTIVITIES];

	extern __shared__ uint8_t dynamicSharedMemory[];
	if (threadIdx.x == 0)	{
		/* SET VARIABLES */
		iter = 0;
		initialized = false;
		blockTabuIdx = 0;
		blockWriteBestBlock = false;
		blockReadSetSolution = false;
		blockWriteSetSolution = false;
		blockCriticalPathLengthAchieved= false;
		blockNumberOfIterationsSinceBest = 0;
		blockIndexOfSetSolution = blockIdx.x % cudaData.totalSolutions;
		maximalNeighbourhoodSize = (cudaData.numberOfActivities-2)*cudaData.swapRange;
		blockReorderingArray = cudaData.swapMergeArray+blockIdx.x*maximalNeighbourhoodSize;
		blockReorderingArrayHelp = cudaData.mergeHelpArray+blockIdx.x*maximalNeighbourhoodSize;
		blockTabuList = cudaData.tabuLists+blockIdx.x*cudaData.maxTabuListSize;
		blockTabuListSize = cudaData.maxTabuListSize-((cudaData.maxTabuListSize*blockIdx.x)/(4*gridDim.x));
		blockTabuCache = cudaData.tabuCaches+blockIdx.x*cudaData.numberOfActivities*cudaData.numberOfActivities;
		blockBestSolution = cudaData.blocksBestSolution+blockIdx.x*cudaData.numberOfActivities;

		hiprand_init(3*blockIdx.x+71, blockIdx.x, 0, &randState);
		blockMaximalNumberOfIterationsSinceBest = hiprand(&randState) % cudaData.maximalIterationsSinceBest;
		
		/* ASSIGN SHARED MEMORY */
		blockMergeArray = (MoveInfo*) dynamicSharedMemory; 
		blockAddedEdges = (Edge*) (blockMergeArray+blockDim.x);
		if (maximalNeighbourhoodSize < 0xffff)	{
			blockPartitionCounterUInt16 = (uint16_t*) (blockAddedEdges+cudaData.numberOfAddedEdges);
			blockPartitionCounterUInt32 = NULL;
			blockCurrentOrder = blockPartitionCounterUInt16+blockDim.x;
		} else	{
			blockPartitionCounterUInt32 = (uint32_t*) (blockAddedEdges+cudaData.numberOfAddedEdges);
			blockPartitionCounterUInt16 = NULL;
			blockCurrentOrder = (uint16_t*) (blockPartitionCounterUInt32+blockDim.x);
		}	
		blockResourceIndices = blockCurrentOrder+cudaData.numberOfActivities;
		blockActivitiesDuration = (uint8_t*) (blockResourceIndices+cudaData.numberOfResources+1);
		if (cudaData.copySuccessorsMatrixToSharedMemory)
			blockSuccessorsMatrix = blockActivitiesDuration+cudaData.numberOfActivities;
		else
			blockSuccessorsMatrix = cudaData.successorsMatrix;
	}
	__syncthreads();

	for (uint32_t i = threadIdx.x; i < cudaData.numberOfResources+1; i += blockDim.x)	{
		blockResourceIndices[i] = cudaData.resourceIndices[i];
	}

	for (uint32_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)	{
		blockActivitiesDuration[i] = cudaData.durationOfActivities[i];
	}

	for (uint32_t i = threadIdx.x; i < cudaData.numberOfAddedEdges; i += blockDim.x)	{
		blockAddedEdges[i] = cudaData.addedEdges[blockIndexOfSetSolution*cudaData.numberOfAddedEdges+i];
	}

	if (cudaData.copySuccessorsMatrixToSharedMemory)	{
		for (uint32_t i = threadIdx.x; i < cudaData.successorsMatrixSize; i += blockDim.x)
			blockSuccessorsMatrix[i] = cudaData.successorsMatrix[i];
	}

	// Block have to obtain initial read access.
	if (threadIdx.x == 0)	{
		while (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) != DATA_AVAILABLE)
			;
		blockBestCost = cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost;
	}
	__syncthreads();

	// Copy solution from a set of solutions to local block order.
	for (uint32_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)	{
		blockCurrentOrder[i] = cudaData.ordersOfSolutions[blockIndexOfSetSolution*cudaData.numberOfActivities+i];
	}
	__syncthreads();

	// Free read lock.
	if (threadIdx.x == 0)	{
		atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
	}

	while (iter < cudaData.numberOfIterationsPerBlock && !blockCriticalPathLengthAchieved)	{

		for (uint16_t i = threadIdx.x+1; i < (cudaData.numberOfActivities-1); i += blockDim.x)	{
			bool relationsBroken = false;
			struct MoveIndices info;
			for (uint16_t j = i+1; j < i+1+cudaData.swapRange; ++j)	{
				info.i = info.j = 0;
				if ((i < cudaData.numberOfActivities-2) && (j < cudaData.numberOfActivities-1) && !relationsBroken)	{
					if (cudaGetMatrixBit(blockSuccessorsMatrix, cudaData.numberOfActivities, blockCurrentOrder[i], blockCurrentOrder[j]) == false)	{
						info.i = i; info.j = j;
					}	else	{
						relationsBroken = true;
					}
				}
				blockReorderingArray[(i-1)*cudaData.swapRange+(j-1-i)] = info;
			}
		}
		__syncthreads();

		uint32_t swapMoves = 0;
		if (blockPartitionCounterUInt16 != NULL)
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArray, (uint32_t*) blockReorderingArrayHelp,  blockPartitionCounterUInt16, maximalNeighbourhoodSize);
		else
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArray, (uint32_t*) blockReorderingArrayHelp,  blockPartitionCounterUInt32, maximalNeighbourhoodSize);

		for (uint32_t i = threadIdx.x; i < swapMoves; i += blockDim.x)	{
			struct MoveIndices *move = &blockReorderingArrayHelp[i];
			if (cudaCheckSwapPrecedencePenalty(blockCurrentOrder, blockSuccessorsMatrix, cudaData.numberOfActivities, move->i, move->j, true) == false)	{
				move->i = move->j = 0;
			}
		}
		__syncthreads();
		
		if (blockPartitionCounterUInt16 != NULL)
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArrayHelp, (uint32_t*) blockReorderingArray,  blockPartitionCounterUInt16, swapMoves);
		else
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArrayHelp, (uint32_t*) blockReorderingArray,  blockPartitionCounterUInt32, swapMoves);

		blockMergeArray[threadIdx.x].cost = 0xffffffff;
		for (uint32_t i = threadIdx.x; i < swapMoves; i += blockDim.x)	{
			struct MoveIndices *move = &blockReorderingArray[i];
			uint32_t threadBestCost = blockMergeArray[threadIdx.x].cost;
			uint32_t totalEval = cudaEvaluateOrder(cudaData, blockCurrentOrder, move->i, move->j, blockActivitiesDuration, blockResourceIndices, threadResourcesLoad,
					threadStartValues, threadRemainingResourcesCapacity, threadStartTimesById, cudaData.capacityResolutionAlgorithm);
			totalEval += cudaComputeTardinessPenalty(cudaData.numberOfActivities, blockActivitiesDuration, blockBestCost-1, threadStartTimesById);
			totalEval += cudaComputePenaltyOfEdgeViolations(cudaData, blockAddedEdges, threadStartTimesById);
			totalEval = (totalEval > 0x0000ffff ? 0xffff0000 : totalEval<<16);
			totalEval |= (hiprand(&threadRandState) & 0x0000ffff);
			uint32_t precedencePenalty = cudaComputePrecedencePenalty(cudaData.numberOfActivities, blockSuccessorsMatrix, blockActivitiesDuration, threadStartTimesById);
			if (precedencePenalty > 0)	{
				printf("ERROR: block %d, thread %d, infeasible solution!\n", blockIdx.x, threadIdx.x);
			}
			bool isPossibleMove = cudaIsPossibleMove(cudaData.numberOfActivities, move->i, move->j, blockTabuCache);
			if ((isPossibleMove && totalEval < threadBestCost) || (totalEval>>16) < blockBestCost)	{
				struct MoveInfo newBestThreadSolution = { move->i, move->j, totalEval };
				blockMergeArray[threadIdx.x] = newBestThreadSolution;
			}
		}
		__syncthreads();

		for (uint16_t k = blockDim.x/2; k > 0; k >>= 1)	{
			if (threadIdx.x < k)	{
				if (blockMergeArray[threadIdx.x].cost > blockMergeArray[threadIdx.x+k].cost)
					blockMergeArray[threadIdx.x] = blockMergeArray[threadIdx.x+k];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0)	{
			blockReadPossible = false;
			iterBestMove = blockMergeArray[0];
			iterBestMove.cost >>= 16;
			atomicAdd((unsigned long long*) cudaData.evaluatedSchedules, swapMoves);
			atomicInc(&cudaData.infoAboutSolutions[blockIndexOfSetSolution].iterationCounter, 0xffffffff);
			if (iterBestMove.cost < blockBestCost)	{
				blockWriteBestBlock = true;
				blockBestCost = iterBestMove.cost;
				blockNumberOfIterationsSinceBest = 0;
			}

			uint32_t readSlotCost = cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost;
			if (blockNumberOfIterationsSinceBest >= blockMaximalNumberOfIterationsSinceBest || readSlotCost != blockBestCost || *cudaData.bestSolutionCost == cudaData.criticalPathLength) {
				bool setOfSolutionsAccess = false;
				if (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					setOfSolutionsAccess = true;

				if (setOfSolutionsAccess)	{

					if (blockBestCost < cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost)	{
						blockWriteSetSolution = true;
						cudaData.infoAboutSolutions[blockIndexOfSetSolution].readCounter = 0;
						cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost = blockBestCost;
						if (blockBestCost < *cudaData.bestSolutionCost)	{
							*cudaData.bestSolutionCost =  blockBestCost;
							*cudaData.indexToTheBestSolution = blockIndexOfSetSolution;
						}
					}	else	{
						atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
					}

					if (*cudaData.bestSolutionCost == cudaData.criticalPathLength)	{
						blockCriticalPathLengthAchieved = true;
					}

					if (readSlotCost < blockBestCost || blockNumberOfIterationsSinceBest >= blockMaximalNumberOfIterationsSinceBest)	{
						blockReadSetSolution = true;
					}
				}
			}  else if (!blockWriteBestBlock)	{
				++blockNumberOfIterationsSinceBest;
			}
		}
		
		if (blockMergeArray[0].cost == 0xffffffff)	{
			// Empty expanded neighborhood. Tabu list will be pruned.
			cudaClearTabuList(cudaData.numberOfActivities, blockTabuList, blockTabuCache, blockTabuListSize/3);
		} else if (threadIdx.x == 0)	{
			// Apply best move.
			uint16_t t = blockCurrentOrder[iterBestMove.i];
			blockCurrentOrder[iterBestMove.i] = blockCurrentOrder[iterBestMove.j];
			blockCurrentOrder[iterBestMove.j] = t;
			// Add move to tabu list.
			cudaAddTurnToTabuList(cudaData.numberOfActivities, iterBestMove.i, iterBestMove.j, blockTabuList, blockTabuCache, blockTabuIdx, blockTabuListSize);
		}
		__syncthreads();

		if (blockWriteBestBlock == true)	{
			if (threadIdx.x == 0)	{
				uint16_t improvedCost = cudaShakingDownEvaluation(cudaData, blockCurrentOrder, threadStartTimesById, blockActivitiesDuration, 
						blockResourceIndices, threadResourcesLoad, threadStartValues, threadRemainingResourcesCapacity);
				if (improvedCost < blockBestCost)	{
					blockBestCost = improvedCost;
					cudaConvertStartTimesById2ActivitiesOrder(blockCurrentOrder, threadStartTimesById, cudaData.numberOfActivities);
				}
			}
			__syncthreads();
			for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
				blockBestSolution[i] = blockCurrentOrder[i];
			blockWriteBestBlock = false; initialized = true;
		}
		__syncthreads();

		if (blockWriteSetSolution == true)	{
			if (!initialized)
				printf("ERROR - write not-initialized solution!!\n");
			for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
				cudaData.ordersOfSolutions[blockIndexOfSetSolution*cudaData.numberOfActivities+i] = blockBestSolution[i];
			for (uint16_t i = threadIdx.x; i < cudaData.maxTabuListSize; i += blockDim.x)
				cudaData.tabuListsOfSetOfSolutions[blockIndexOfSetSolution*cudaData.maxTabuListSize+i] = blockTabuList[i];
			__threadfence();
			__syncthreads();
			if (threadIdx.x == 0)	{
				blockWriteSetSolution = false; 
				atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
			}
		}

		if (blockReadSetSolution == true)	{
			if (threadIdx.x == 0)	{
				if (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					blockReadPossible = true;
			}
			__syncthreads();
			if (blockReadPossible)	{
				if (threadIdx.x == 0)	{
					blockIndexOfSetSolution = (blockIndexOfSetSolution+1) % cudaData.totalSolutions;
				}
				__syncthreads();
				// Read solution from a set to block memory.
				cudaReadExternalSolution(cudaData.numberOfActivities, blockTabuList, blockTabuCache, blockTabuListSize, blockCurrentOrder,
						cudaData.ordersOfSolutions+blockIndexOfSetSolution*cudaData.numberOfActivities, cudaData.tabuListsOfSetOfSolutions+blockIndexOfSetSolution*cudaData.maxTabuListSize);
				if (threadIdx.x == 0)	{
					blockBestCost = cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost;
					uint32_t readCounter = ++cudaData.infoAboutSolutions[blockIndexOfSetSolution].readCounter;
					blockNumberOfIterationsSinceBest = 0;

					blockReadSetSolution = false; initialized = false;
					blockMaximalNumberOfIterationsSinceBest = hiprand(&randState) % cudaData.maximalIterationsSinceBest;
					atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
					if (readCounter > cudaData.maximalValueOfReadCounter)
						cudaDiversificationOfSolution(cudaData.numberOfActivities, blockCurrentOrder, blockSuccessorsMatrix, cudaData.numberOfDiversificationSwaps, &randState);
				}
			}
		}

		if (threadIdx.x == 0)	{
			++iter;
		}
		__syncthreads();
	}

	// Write solution if is better than the best found.
	if (threadIdx.x == 0)	{
		while (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) != DATA_AVAILABLE)
			;
	}
	__syncthreads();

	if (*cudaData.bestSolutionCost > blockBestCost)	{
		if (threadIdx.x == 0 && !initialized)	{
			printf("ERROR - write not-initialized solution!!\n");
			printf("original %d; new %d\n", *cudaData.bestSolutionCost, blockBestCost);
		}
		for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
			cudaData.ordersOfSolutions[blockIndexOfSetSolution*cudaData.numberOfActivities+i] = blockBestSolution[i];
		for (uint16_t i = threadIdx.x; i < cudaData.maxTabuListSize; i += blockDim.x)
			cudaData.tabuListsOfSetOfSolutions[blockIndexOfSetSolution*cudaData.maxTabuListSize+i] = blockTabuList[i];
		if (threadIdx.x == 0)	{
			*cudaData.bestSolutionCost = blockBestCost;
			*cudaData.indexToTheBestSolution = blockIndexOfSetSolution;
		}
	}
	__threadfence();
	__syncthreads();

	if (threadIdx.x == 0)
		atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);

	return;
}


/* START MAIN CUDA KERNEL */

void runCudaSolveRCPSP(int numberOfBlock, int numberOfThreadsPerBlock, int computeCapability, int dynSharedMemSize, const CudaData& cudaData)	{
	if (dynSharedMemSize < 7950)	{
		// Prefare 16 kB shared memory + 48 kB cache L1.
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaSolveRCPSP), hipFuncCachePreferL1);
	} else {
		// Prefare 48 kB shared memory + 16 kB cache L1.
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaSolveRCPSP), hipFuncCachePreferShared);
	}
	// Set maximum amount of dynamic memory to 1 MB.
	hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024);
	// Launch the main GPU kernel.
	cudaSolveRCPSP<<<numberOfBlock,numberOfThreadsPerBlock,dynSharedMemSize>>>(cudaData);
	hipDeviceSynchronize();
}

