#include "hip/hip_runtime.h"
/*!
 * \file CudaFunctions.cu
 * \author Libor Bukata
 * \brief RCPSP Cuda functions.
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "CudaConstants.h"
#include "CudaFunctions.cuh"

#if defined _WIN32 || defined _WIN64 || defined WIN32 || defined WIN64
#include "hip/hip_runtime.h"
#include ""
#endif

using std::cerr;
using std::cout;
using std::endl;

//! Texture reference of activities resource requirements.
texture<uint8_t,1,hipReadModeElementType> cudaActivitiesResourcesTex;
//! Texture reference of predecessors.
texture<uint16_t,1,hipReadModeElementType> cudaPredecessorsTex;
//! Texture reference of predecessors indices.
texture<uint16_t,1,hipReadModeElementType> cudaPredecessorsIndicesTex;
//! Texture reference of successors.
texture<uint16_t,1,hipReadModeElementType> cudaSuccessorsTex;
//! Texture reference of successors indices.
texture<uint16_t,1,hipReadModeElementType> cudaSuccessorsIndicesTex;

//! The longest paths from the end dummy activity to the others in the transformed graph.
__constant__ uint16_t rightLeftLongestPaths[NUMBER_OF_ACTIVITIES];


/* CUDA BIND TEXTURES */

int bindTexture(void *texData, int32_t arrayLength, int option)	{
	switch (option)	{
		case ACTIVITIES_RESOURCES:
			return hipBindTexture(NULL, cudaActivitiesResourcesTex, texData, arrayLength*sizeof(uint8_t));
		case PREDECESSORS:
			return hipBindTexture(NULL, cudaPredecessorsTex, texData, arrayLength*sizeof(uint16_t));
		case PREDECESSORS_INDICES:
			return hipBindTexture(NULL, cudaPredecessorsIndicesTex, texData, arrayLength*sizeof(uint16_t));
		case SUCCESSORS:
			return hipBindTexture(NULL, cudaSuccessorsTex, texData, arrayLength*sizeof(uint16_t));
		case SUCCESSORS_INDICES:
			return hipBindTexture(NULL, cudaSuccessorsIndicesTex, texData, arrayLength*sizeof(uint16_t));
		default:
			cerr<<"bindTextures: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}

int unbindTexture(int option)	{
	switch (option)	{
		case ACTIVITIES_RESOURCES:
			return hipUnbindTexture(cudaActivitiesResourcesTex);
		case PREDECESSORS:
			return hipUnbindTexture(cudaPredecessorsTex);
		case PREDECESSORS_INDICES:
			return hipUnbindTexture(cudaPredecessorsIndicesTex);
		case SUCCESSORS:
			return hipUnbindTexture(cudaSuccessorsTex);
		case SUCCESSORS_INDICES:
			return hipUnbindTexture(cudaSuccessorsIndicesTex);
		default:
			cerr<<"unbindTextures: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}

int memcpyToSymbol(void *source, int32_t arrayLength, int option)	{
	switch (option)	{
		case THE_LONGEST_PATHS:
			return hipMemcpyToSymbol((const char*) source, (void*) rightLeftLongestPaths, arrayLength*sizeof(uint16_t));
		default:
			cerr<<"memcpyToSymbol: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}


/* CUDA IMPLEMENT OF SOURCES LOAD - CAPACITY RESOLUTION  */

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \brief Prepare arrays for next use (schedule evaluation).
 */
inline __device__ void cudaPrepareArrays(const CudaData& cudaData, uint16_t *& resourcesLoad, uint16_t *& startValues)	{
	for (uint16_t i = 0; i < cudaData.sumOfCapacities; ++i)
		resourcesLoad[i] = 0;
	for (uint16_t i = 0; i < cudaData.maximalCapacityOfResource; ++i)
		startValues[i] = 0;
}

/*!
 * \param numberOfResources Number of resources.
 * \param activityId Activity identification.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param resourceIndices Access indices for resources.
 * \return Earliest start time of an activity.
 * \brief Function return earliest possible start time of an activity. Precedence relations are ignored.
 */
inline __device__ uint16_t cudaGetEarliestStartTime(const uint16_t& numberOfResources, const uint16_t& activityId, uint16_t *&resourcesLoad, uint16_t *&resourceIndices) {
	uint16_t bestStart = 0;
	for (uint8_t resourceId = 0; resourceId < numberOfResources; ++resourceId)	{
		uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		if (activityRequirement > 0)
			bestStart = max(resourcesLoad[resourceIndices[resourceId+1]-activityRequirement], bestStart);
	}
	return bestStart;
}

/*!
 * \param activityId Activity identification.
 * \param activityStart Start time of an activity.
 * \param activityStop Stop time of an activity.
 * \param numberOfResources Number of resources.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \brief Function add new activity and update resources arrays. Irreversible process.
 */
inline __device__ void cudaAddActivity(const uint16_t& activityId, const uint16_t& activityStart, const uint16_t& activityStop,
		const uint16_t& numberOfResources, uint16_t *&resourceIndices,  uint16_t *&resourcesLoad, uint16_t *&startValues)	{
	
	int32_t requiredSquares, timeDiff;
	int32_t c, k, capacityOfResource, resourceRequirement, newStartTime, resourceStartIdx;
	for (uint8_t resourceId = 0; resourceId < numberOfResources; ++resourceId)	{
		resourceStartIdx = resourceIndices[resourceId];
		capacityOfResource = resourceIndices[resourceId+1]-resourceStartIdx;
		resourceRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		requiredSquares = resourceRequirement*(activityStop-activityStart);
		if (requiredSquares > 0)	{
			c = 0; k = 0;
			newStartTime = activityStop;
			while (requiredSquares > 0 && k < capacityOfResource)	{
				if (resourcesLoad[resourceStartIdx+k] < newStartTime)	{
					if (c >= resourceRequirement)
						newStartTime = startValues[c-resourceRequirement];
					timeDiff = newStartTime-max(resourcesLoad[resourceStartIdx+k], activityStart);
					if (requiredSquares-timeDiff > 0)	{
						requiredSquares -= timeDiff;
						startValues[c++] = resourcesLoad[resourceStartIdx+k];
						resourcesLoad[resourceStartIdx+k] = newStartTime;
					} else {
						resourcesLoad[resourceStartIdx+k] = newStartTime-timeDiff+requiredSquares;
						break;
					}
				}
				++k;
			}
		}
	}
}

/* CUDA IMPLEMENT OF SOURCES LOAD - TIME RESOLUTION  */

/*!
 * \param numberOfActivities Number of activities in the project.
 * \param numberOfResources Number of renewable resources in the project.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param resourceIndices Access indices for resources.
 * \brief It initializes vectors of free capacities to initial values (capacities of resources).
 */
inline __device__ void cudaPrepareArrays(const uint16_t& numberOfActivities, const uint16_t& numberOfResources, const uint32_t& UBTime,
	       	uint8_t *& remainingResourcesCapacity, uint16_t *& resourceIndices)	{
	for (uint16_t resourceId = 0; resourceId < numberOfResources; ++resourceId)
		for (uint32_t t = 0; t < UBTime; ++t)
			remainingResourcesCapacity[resourceId*UBTime+t] = resourceIndices[resourceId+1]-resourceIndices[resourceId];
}

/*!
 * \param numberOfResources Number of renewable resources in the project.
 * \param activityId Identification of the activity that should be added (required for texture memory access).
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param precTime The earliest precedence violation free start time of the activity activityId.
 * \param activityDuration Duration of the activity activityId.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \return The earliest start time of the activity without resource overload.
 * \brief It finds out the earliest start time of the activity activityId.
 */
inline __device__ uint16_t cudaGetEarliestStartTime(const uint16_t& numberOfResources, const uint16_t& activityId,
		uint8_t *&remainingResourcesCapacity, const uint16_t& precTime, int32_t activityDuration, const uint32_t& UBTime) {
	int32_t loadTime = 0, t = UBTime;
	for (t = precTime; t < UBTime && loadTime < activityDuration; ++t)       {
		bool capacityAvailable = true;
		for (int32_t resourceId = 0; resourceId < numberOfResources && capacityAvailable; ++resourceId)        {
			uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
			if (remainingResourcesCapacity[resourceId*UBTime+t] < activityRequirement)	{
				loadTime = 0;
				capacityAvailable = false;
			}
		}
		if (capacityAvailable == true)
			++loadTime;
	}
	return (uint16_t) t-loadTime;
}

/*!
 * \param activityId Identification of the added activity.
 * \param activityStart Scheduled start time of the activity.
 * \param activityStop Scheduled finish time of the activity.
 * \param numberOfResources Number of renewable resources in the project.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \brief It updates the state of all resources after activity is added.
 */
inline __device__ void cudaAddActivity(const uint16_t& activityId, const uint16_t& activityStart, const uint16_t& activityStop,
		const uint16_t& numberOfResources, uint8_t *&remainingResourcesCapacity, const uint32_t& UBTime)	{
	for (int32_t resourceId = 0; resourceId < numberOfResources; ++resourceId)     {
		uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		for (uint32_t t = activityStart; t < activityStop; ++t)
			remainingResourcesCapacity[resourceId*UBTime+t] -= activityRequirement;
	}
}

/* CUDA IMPLEMENTATION OF THE BASE RESOURCE EVALUATION FUNCTIONS */

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param blockOrder Current order of the activities.
 * \param indexI Swap index i.
 * \param indexJ Swap index j.
 * \param activitiesDuration Duration of the activities.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param startTimesWriterById Array of start times of the scheduled activities ordered by ID's.
 * \param capacityResolution If true then capacity based algorithm is selected else time based algorithm is selected.
 * \param forward It determines if schedule is forward or backward evaluated.
 * \return Schedule length without any penalties.
 * \brief Function evaluate schedule and return total schedule length.
 */
__device__ uint16_t cudaEvaluateOrder(const CudaData& cudaData, uint16_t *&blockOrder, const uint16_t& indexI, const uint16_t& indexJ,
	       	uint8_t *&activitiesDuration,uint16_t *&resourceIndices, uint16_t *resourcesLoad, uint16_t *startValues,
	       	uint8_t *remainingResourcesCapacity, uint16_t *startTimesWriterById, bool capacityResolution, bool forward = true)	{
	// Current cost of the schedule.
	uint16_t scheduleLength = 0;

	// Init state of resources.
	if (capacityResolution == true)
		cudaPrepareArrays(cudaData, resourcesLoad, startValues);
	else
		cudaPrepareArrays(cudaData.numberOfActivities, cudaData.numberOfResources, MAXIMAL_SUM_OF_FLOATS,
				remainingResourcesCapacity, resourceIndices);
	
	for (uint16_t i = 0; i < cudaData.numberOfActivities; ++i)	{

		uint16_t index = ((forward == true) ? i : cudaData.numberOfActivities-i-1);
		uint16_t activityId = blockOrder[index];

		// Logical swap.
		if (index == indexI)
			activityId = blockOrder[indexJ];

		if (index == indexJ)
			activityId = blockOrder[indexI];

		// Get the earliest start time without precedence penalty. (if moves are precedence penalty free)
		uint16_t start = 0;
		uint16_t baseIndex;
		uint16_t numberOfRelatedActivities;
		if (forward == true) {
			baseIndex = tex1Dfetch(cudaPredecessorsIndicesTex, activityId);
			numberOfRelatedActivities = tex1Dfetch(cudaPredecessorsIndicesTex, activityId+1)-baseIndex;
		} else	{
			baseIndex = tex1Dfetch(cudaSuccessorsIndicesTex, activityId);
			numberOfRelatedActivities = tex1Dfetch(cudaSuccessorsIndicesTex, activityId+1)-baseIndex;
		}
		for (uint16_t j = 0; j < numberOfRelatedActivities; ++j)	{
			uint16_t relatedActivityId;
			if (forward == true)
				relatedActivityId = tex1Dfetch(cudaPredecessorsTex, baseIndex+j);
			else
				relatedActivityId = tex1Dfetch(cudaSuccessorsTex, baseIndex+j);
			start = max(startTimesWriterById[relatedActivityId]+activitiesDuration[relatedActivityId], start);
		}

		// Get the earliest start time if the resources restrictions are counted.
		if (capacityResolution == true)
			start = max(cudaGetEarliestStartTime(cudaData.numberOfResources, activityId, resourcesLoad, resourceIndices), start);
		else
			start = max(cudaGetEarliestStartTime(cudaData.numberOfResources, activityId, remainingResourcesCapacity,
						start, activitiesDuration[activityId], MAXIMAL_SUM_OF_FLOATS), start);

		// Add activity = update resources arrays + write start time.
		uint16_t stop = start+activitiesDuration[activityId];
		if (capacityResolution == true)
			cudaAddActivity(activityId, start, stop, cudaData.numberOfResources, resourceIndices, resourcesLoad, startValues);
		else
			cudaAddActivity(activityId, start, stop, cudaData.numberOfResources, remainingResourcesCapacity, MAXIMAL_SUM_OF_FLOATS);
		scheduleLength = max(scheduleLength, stop);

		startTimesWriterById[activityId] = start;
	}

	return scheduleLength;
}

/*!
 * \param order Order of activities.
 * \param timeValuesById Time values of activities. Accessed through the identifications of activities.
 * \param size Length of the order and timeValuesById arrays.
 * \brief It reorders input order in accordance with timeValuesById array. It's stable sort with algorithm complexity O(n^2).
 */
inline __device__ void cudaInsertSort(uint16_t* order, const uint16_t * const& timeValuesById, const int16_t& size)	{
	for (int32_t i = 1; i < size; ++i)	{
		for (int32_t j = i; (j > 0) && ((timeValuesById[order[j]] < timeValuesById[order[j-1]]) == true); --j)	{
			uint16_t t = order[j];
			order[j] = order[j-1];
			order[j-1] = t;
		}
	}
}

/*!
 * \param order Order of activities.
 * \param startTimesById Start time values of activities. Accessed through the identifications of activities.
 * \param size Length of the order and timeValuesById arrays.
 * \brief It converts startTimesById array to activities order.
 */
inline __device__ void cudaConvertStartTimesById2ActivitiesOrder(uint16_t *& order, uint16_t *startTimesById, uint16_t size)	{
	cudaInsertSort(order, startTimesById, size);
}

/* CHECK PRECEDENCE FUNCTIONS */

/*!
 * \param successorsMatrix Bit matrix of successors.
 * \param numberOfActivities Number of activities.
 * \param activityId1 Activity identification.
 * \param activityId2 Activity identification.
 * \return True if an activity with identification activityId2 is successor of an activity with identification activityId1.
 * \brief Check if activity ID2 is successor of activity ID1.
 */
inline __device__ bool cudaGetMatrixBit(const uint8_t * const& successorsMatrix, const uint16_t& numberOfActivities, const int16_t& activityId1, const int16_t& activityId2)	{
	uint32_t bitPossition = activityId1*numberOfActivities+activityId2;
	if ((successorsMatrix[bitPossition/8] & (1<<(bitPossition % 8))) > 0)
		return true;
	else
		return false;
}

/*!
 * \param order Sequence of activities.
 * \param successorsMatrix Bit matrix of successors.
 * \param numberOfActivities Number of activities.
 * \param i Index i of swap.
 * \param j Index j of swap.
 * \param light If true then light version is executed. (precedences from activity at index i aren't checked)
 * \return True if current swap won't break relation precedences else false.
 * \brief Check if requested move is precedence penalty free.
 */
__device__ bool cudaCheckSwapPrecedencePenalty(const uint16_t * const& order, const uint8_t * const& successorsMatrix, const uint16_t& numberOfActivities, int16_t i, int16_t j, bool light = false)	{
	if (i > j)	{
		int16_t t = i;
		i = j; j = t;
	}
	for (uint16_t k = i; k < j; ++k)	{
		if (cudaGetMatrixBit(successorsMatrix, numberOfActivities, order[k], order[j]) == true)
			return false;
	}
	if (!light)	{
		for (uint16_t k = i+1; k <= j; ++k)	{
			if (cudaGetMatrixBit(successorsMatrix, numberOfActivities, order[i], order[k]) == true)
				return false;
		}
	}
	return true;
}

/*!
 * \param numAct The number of activities.
 * \param successorsMatrix Binary matrix of successors.
 * \param activitiesDuration Duration of each activity.
 * \param startTimesById Array of start time values of the scheduled activities ordered by ID's.
 * \return The precedence penalty.
 * \brief It finds out all precedence penalties and computes penalty.
 * \note The penalty should be zero since only non-precedence breaking moves are allowed.
 */
__device__ uint32_t cudaComputePrecedencePenalty(uint16_t numAct, uint8_t *successorsMatrix, uint8_t *activitiesDuration, uint16_t *startTimesById)  {
	uint32_t penalty = 0;
	for (uint16_t id1 = 0; id1 < numAct; ++id1)        {
		for (uint16_t id2 = 0; id2 < numAct; ++id2)        {
			if (id1 != id2 && cudaGetMatrixBit(successorsMatrix, numAct, id1, id2) == true)	{
				if (startTimesById[id1]+activitiesDuration[id1] > startTimesById[id2])
					penalty += startTimesById[id1]+activitiesDuration[id1]-startTimesById[id2];
			}

		}
	}
	return penalty;
}

/*!
 * \param numberOfActivities The number of the activities in the project.
 * \param activitiesDuration Duration of each activity.
 * \param makespan The best known project makespan.
 * \param startTimesById Array of start time values of the scheduled activities ordered by ID's.
 * \return It returns overall tardiness penalty.
 */
__device__ uint32_t cudaComputeTardinessPenalty(uint16_t numberOfActivities, uint8_t *activitiesDuration, uint32_t makespan, uint16_t *startTimesById)	{
	uint32_t overhangPenalty = 0;
	for (uint16_t id = 0; id < numberOfActivities; ++id)	{
		if (startTimesById[id]+activitiesDuration[id]+rightLeftLongestPaths[id] > makespan)
			overhangPenalty += startTimesById[id]+activitiesDuration[id]+rightLeftLongestPaths[id]-makespan;
	}
	return overhangPenalty;
}

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param blockOrder Order of activities.
 * \param bestScheduleStartTimesById Start time values of activities for the best shaked schedule.
 * \param activitiesDuration Duration of each activity.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \return The makespan of the best found shaked schedule.
 * \brief Iterative method tries to shake down activities in the schedule to ensure equally loaded resources. 
 * Therefore, the shorter schedule could be found.
 */
__device__ uint16_t cudaShakingDownEvaluation(const CudaData& cudaData, uint16_t *& blockOrder,
	       	uint16_t *bestScheduleStartTimesById, uint8_t *& activitiesDuration, uint16_t *& resourceIndices,
		uint16_t *resourcesLoad, uint16_t *startValues, uint8_t *remainingResourcesCapacity)	{
	int32_t scheduleLength = 0;
	uint16_t bestScheduleLength = 0xffff;
	uint16_t *currentOrder = new uint16_t[cudaData.numberOfActivities];
	if (!currentOrder)
		return bestScheduleLength;
	uint16_t *timeValuesById = new uint16_t[cudaData.numberOfActivities];
	if (!timeValuesById)	{
		delete[] currentOrder;
		return bestScheduleLength;
	}

	for (uint16_t i = 0; i < cudaData.numberOfActivities; ++i)
		currentOrder[i] = blockOrder[i];

	while (true)	{
		scheduleLength = cudaEvaluateOrder(cudaData, currentOrder, 0xffff, 0xffff, activitiesDuration, resourceIndices,
			       	resourcesLoad, startValues, remainingResourcesCapacity, timeValuesById, false, true);

		if (scheduleLength < bestScheduleLength)	{
			bestScheduleLength = scheduleLength;
			if (bestScheduleStartTimesById != NULL)	{
				for (uint16_t id = 0; id < cudaData.numberOfActivities; ++id)
					bestScheduleStartTimesById[id] = timeValuesById[id];
			}
		} else {
			break;
		}

		for (uint16_t id = 0; id < cudaData.numberOfActivities; ++id)
			timeValuesById[id] += activitiesDuration[id];

		cudaInsertSort(currentOrder, timeValuesById, cudaData.numberOfActivities);

		int32_t scheduleLengthBackward = cudaEvaluateOrder(cudaData, currentOrder, 0xffff, 0xffff, activitiesDuration,
			       	resourceIndices, resourcesLoad, startValues, remainingResourcesCapacity, timeValuesById, false, false);
		int32_t diffCmax = scheduleLength-scheduleLengthBackward;

		for (uint32_t id = 0; id < cudaData.numberOfActivities; ++id)
			timeValuesById[id] = scheduleLengthBackward-timeValuesById[id]-activitiesDuration[id];

		for (uint32_t id = 0; id < cudaData.numberOfActivities; ++id)	{
			if (((int32_t) timeValuesById[id])+diffCmax > 0)
				timeValuesById[id] += diffCmax;
			else
				timeValuesById[id] = 0;
		}

		cudaInsertSort(currentOrder, timeValuesById, cudaData.numberOfActivities);
	}

	delete[] timeValuesById;
	delete[] currentOrder;

	return bestScheduleLength;
}


/* HASH TABLE INDEX FUNCTION */

/*!
 * \param numAct Number of activities.
 * \param cudaBlockOrder Current order of the block.
 * \param actX Swap index i - logical swap.
 * \param actY Swap index j - logical swap.
 * \param actI Swap index i - store purpose.
 * \param actJ Swap index j - store purpose.
 * \return Index to a hash table.
 * \brief Function compute hash table index for tabu hash purposes.
 */
__device__ uint32_t cudaComputeHashTableIndex(uint16_t numAct, uint16_t *cudaBlockOrder, uint16_t actX, uint16_t actY, uint32_t actI, uint32_t actJ)	{
	uint32_t hashValue = 1;

	hashValue *= (R+2*actI);
	hashValue ^= actI;

	for (uint32_t i = 1; i < numAct-1; ++i)	{
		uint32_t activityId = cudaBlockOrder[i];
		if (i == actX)
			activityId = cudaBlockOrder[actY];
		if (i == actY)
			activityId = cudaBlockOrder[actX];

		hashValue *= (R+2*activityId*i);
		hashValue ^= activityId;
	}

	hashValue *= (R+2*actJ);
	hashValue ^= actJ;

	hashValue /= 2;
	hashValue &= 0x00ffffff;	// Size of the hash table is 2^24.

	return hashValue;
}

/*	CUDA IMPLEMENT OF SIMPLE TABU LIST */

/*!
 * \param numberOfActivities Number of activities.
 * \param i Swap index i.
 * \param j Swap index j.
 * \param tabuCache Block tabu cache - fast check if move is in tabu list.
 * \return True if move is possible else false.
 * \brief Check if move is in tabu list.
 */
inline __device__ bool cudaIsPossibleMove(const uint16_t& numberOfActivities, const uint16_t& i, const uint16_t& j, uint8_t *&tabuCache)	{
	if (tabuCache[i*numberOfActivities+j] == 0 || tabuCache[j*numberOfActivities+i] == 0)
		return true;
	else
		return false;
}

/*!
 * \param numberOfActivities Number of activities.
 * \param i Swap index i of added move.
 * \param j Swap index j of added move.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param tabuIdx Current index at tabu list.
 * \param tabuListSize Tabu list size.
 * \brief Add specified move to tabu list and update tabu cache.
 */
inline __device__ void cudaAddTurnToTabuList(const uint16_t& numberOfActivities, const uint16_t& i, const uint16_t& j,
		MoveIndices *&tabuList, uint8_t *&tabuCache, uint16_t& tabuIdx, const uint16_t& tabuListSize)	{

	MoveIndices move = tabuList[tabuIdx];
	uint16_t iOld = move.i, jOld = move.j;

	if (iOld != 0 && jOld != 0)
		tabuCache[iOld*numberOfActivities+jOld] = tabuCache[jOld*numberOfActivities+iOld] = 0;

	move.i = i; move.j = j;
	tabuList[tabuIdx] = move;
	tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 1;

	tabuIdx = (tabuIdx+1) % tabuListSize;
}


/* HELP FUNCTIONS */

/*!
 * \param numberOfActivities Number of activities.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param numberOfElements Number of tabu list elements that will be removed.
 * \brief Remove specified number of elements from tabu list and update tabu cache.
 */
inline __device__ void cudaClearTabuList(const uint16_t& numberOfActivities, MoveIndices *tabuList, uint8_t *tabuCache, const uint16_t& numberOfElements)	{
	for (uint16_t k = threadIdx.x; k < numberOfElements; k += blockDim.x)	{
		MoveIndices *tabuMove = &tabuList[k];
		uint16_t i = tabuMove->i, j = tabuMove->j;
		tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 0;
		tabuMove->j = tabuMove->i = 0;
	}
	__syncthreads();
	return;
}

/*!
 * \param numberOfActivities Number of activities.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param tabuListSize Block tabu list size.
 * \param blockOrder Block schedule - order.
 * \param externalSolution Solution from a set or the best global solution. (order)
 * \param externalTabuList Tabu list of external solution.
 * \brief Replace current block solution with a read external solution (order+tabu).
 */
inline __device__ void cudaReadExternalSolution(const uint16_t& numberOfActivities, MoveIndices *tabuList, uint8_t *tabuCache, const uint16_t& tabuListSize,
		uint16_t *blockOrder, uint16_t *externalSolution, MoveIndices *externalTabuList)	{
	// Clear current tabu list and tabu cache.
	cudaClearTabuList(numberOfActivities, tabuList, tabuCache, tabuListSize);
	// Read block order.
	for (uint16_t i = threadIdx.x; i < numberOfActivities; i += blockDim.x)
		blockOrder[i] = externalSolution[i];
	// Read block tabu list and create tabu cache.
	for (uint16_t l = threadIdx.x; l < tabuListSize; l += blockDim.x)	{
		tabuList[l] = externalTabuList[l];
		MoveIndices *move = &tabuList[l];
		uint16_t i = move->i, j = move->j;
		tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 1;
	}
	__syncthreads();
	return;
}

/* REORDER ARRAY FUNCTION */

/*!
 * \tparam T uint16_t or uint32_t.
 * \param moves Array of moves which should be reorder.
 * \param resultMerge Result array of reordered moves.
 * \param threadsCounter Helper array for threads counters.
 * \param size How many elements will be processed at moves array.
 * \return Number of written elements to resultMerge array.
 * \brief Move all valid moves to the resultMerge array and return number of valid moves.
 */
template <typename T>
inline __device__ uint32_t cudaReorderMoves(uint32_t *moves, uint32_t *resultMerge, T *threadsCounter, const uint32_t& size)	{
	threadsCounter[threadIdx.x] = 0;
	uint32_t threadAmount = size/blockDim.x+1;
	for (uint32_t i = threadIdx.x*threadAmount; i < size && i < (threadIdx.x+1)*threadAmount; ++i)	{
		if (moves[i] != 0)
			++threadsCounter[threadIdx.x];
	}
	__syncthreads();
	for (uint32_t k = 0; (1<<k) < blockDim.x; ++k)   {
		uint32_t step = 1<<k;
		uint32_t begIdx = (step-1)+2*step*threadIdx.x;
		if (begIdx < blockDim.x-step)
			threadsCounter[begIdx+step] += threadsCounter[begIdx];
		__syncthreads();
	}
	for (int32_t k = (blockDim.x>>1); k > 1; k >>= 1)	{
		uint32_t step = k/2;
		uint32_t begIdx = (k-1)+2*step*threadIdx.x;
		if (begIdx < blockDim.x-step) 
			threadsCounter[begIdx+step] += threadsCounter[begIdx];
		__syncthreads();
	}
	uint32_t threadStartIndex = threadIdx.x > 0 ? threadsCounter[threadIdx.x-1] : 0;
	for (uint32_t i = threadIdx.x*threadAmount; i < size && i < (threadIdx.x+1)*threadAmount; ++i)	{
		if (moves[i] != 0)
			resultMerge[threadStartIndex++] = moves[i];
	}
	__syncthreads();
	return threadsCounter[blockDim.x-1];
}

/* DIVERSIFICATION FUNCTION */

/*!
 * \param numberOfActivities Number of activities.
 * \param order Current schedule - sequence of activities.
 * \param successorsMatrix Bit matrix of successors.
 * \param diversificationSwaps Number of diversification swaps.
 * \param state State of the random generator.
 * \brief Function performs specified number of precedence penalty free swaps.
 */
inline __device__ void cudaDiversificationOfSolution(const uint16_t& numberOfActivities, uint16_t *order, const uint8_t *successorsMatrix, 
		const uint32_t& diversificationSwaps, hiprandState *state)	{
		
	uint32_t performedSwaps = 0;
	while (performedSwaps < diversificationSwaps)  {
		uint16_t i = (hiprand(state) % (numberOfActivities-2)) + 1;
		uint16_t j = (hiprand(state) % (numberOfActivities-2)) + 1;
		if ((i != j) && (cudaCheckSwapPrecedencePenalty(order, successorsMatrix, numberOfActivities, i, j) == true))	{
			uint16_t t = order[i];
			order[i] = order[j];
			order[j] = t;
			++performedSwaps;
		}
	}
	return;
}


/*	CUDA IMPLEMENT OF GLOBAL KERNEL */

/*!
 * Global function for RCPSP problem. Blocks communicate with each other through global memory.
 * Local variables are coalesced. Dynamic shared memory and texture memory is used.
 * \param cudaData All required constants, pointers to device memory, setting variables, ....
 * \brief Solve RCPSP problem on GPU.
 */
__global__ void cudaSolveRCPSP(const CudaData cudaData)	{
	
	__shared__ uint32_t iter;
	__shared__ MoveInfo iterBestMove;
	__shared__ uint32_t blockBestCost;
	__shared__ uint16_t *blockBestSolution;
	__shared__ uint32_t maximalNeighbourhoodSize;
	__shared__ uint8_t *blockActivitiesDuration;
	__shared__ uint16_t *blockCurrentOrder;
	__shared__ uint8_t *blockSuccessorsMatrix;
	__shared__ MoveInfo *blockMergeArray;
	__shared__ uint16_t *blockPartitionCounterUInt16;
	__shared__ uint32_t *blockPartitionCounterUInt32;
	__shared__ MoveIndices *blockReorderingArray;
	__shared__ MoveIndices *blockReorderingArrayHelp;

	__shared__ uint16_t blockTabuIdx;
	__shared__ uint16_t blockTabuListSize;
	__shared__ MoveIndices *blockTabuList;
	__shared__ uint8_t *blockTabuCache;
	__shared__ int32_t blockIndexOfSetSolution;
	__shared__ bool blockReadPossible;
	__shared__ bool blockReadFromSet;
	__shared__ bool blockWriteBestBlock;
	__shared__ bool blockReadSetSolution;
	__shared__ bool blockWriteSetSolution;
	__shared__ bool blockReadGlobalBestSolution;
	__shared__ bool blockWriteGlobalBestSolution;
	__shared__ bool blockCriticalPathLengthAchieved;
	__shared__ uint32_t blockNumberOfIterationsSinceBest;
	__shared__ uint32_t blockMaximalNumberOfIterationsSinceBest;
	__shared__ uint16_t *blockResourceIndices;

	__shared__ hiprandState randState;

	hiprandState threadRandState;
	hiprand_init(blockDim.x*blockIdx.x+threadIdx.x, threadIdx.x, 0, &threadRandState);

	uint16_t threadResourcesLoad[TOTAL_SUM_OF_CAPACITY];
	uint16_t threadStartValues[MAXIMUM_CAPACITY_OF_RESOURCE];
	uint8_t threadRemainingResourcesCapacity[NUMBER_OF_RESOURCES*MAXIMAL_SUM_OF_FLOATS];
	uint16_t threadStartTimesById[NUMBER_OF_ACTIVITIES];

	extern __shared__ uint8_t dynamicSharedMemory[];
	if (threadIdx.x == 0)	{
		/* SET VARIABLES */
		iter = 0;
		blockTabuIdx = 0;
		blockReadFromSet = true;
		blockWriteBestBlock = false;
		blockReadSetSolution = false;
		blockWriteSetSolution = false;
		blockReadGlobalBestSolution = false;
		blockWriteGlobalBestSolution = false;
		blockCriticalPathLengthAchieved= false;
		blockNumberOfIterationsSinceBest = 0;
		blockIndexOfSetSolution = blockIdx.x % cudaData.solutionsSetSize;
		maximalNeighbourhoodSize = (cudaData.numberOfActivities-2)*cudaData.swapRange;
		blockReorderingArray = cudaData.swapFreeMergeArray+blockIdx.x*maximalNeighbourhoodSize;
		blockReorderingArrayHelp = cudaData.mergeHelpArray+blockIdx.x*maximalNeighbourhoodSize;
		blockTabuList = cudaData.tabuLists+blockIdx.x*cudaData.maxTabuListSize;
		blockTabuListSize = cudaData.maxTabuListSize-((cudaData.maxTabuListSize*blockIdx.x)/(4*gridDim.x));
		blockTabuCache = cudaData.tabuCaches+blockIdx.x*cudaData.numberOfActivities*cudaData.numberOfActivities;
		blockBestSolution = cudaData.blocksBestSolution+blockIdx.x*cudaData.numberOfActivities;

		hiprand_init(3*blockIdx.x+71, blockIdx.x, 0, &randState);
		blockMaximalNumberOfIterationsSinceBest = hiprand(&randState) % cudaData.maximalIterationsSinceBest;
		
		/* ASSIGN SHARED MEMORY */
		blockMergeArray = (MoveInfo*) dynamicSharedMemory; 
		if (maximalNeighbourhoodSize < 0xffff)	{
			blockPartitionCounterUInt16 = (uint16_t*) (blockMergeArray+blockDim.x);
			blockPartitionCounterUInt32 = NULL;
			blockCurrentOrder = blockPartitionCounterUInt16+blockDim.x;
		} else	{
			blockPartitionCounterUInt32 = (uint32_t*) (blockMergeArray+blockDim.x);
			blockPartitionCounterUInt16 = NULL;
			blockCurrentOrder = (uint16_t*) (blockPartitionCounterUInt32+blockDim.x);
		}	
		blockResourceIndices = blockCurrentOrder+cudaData.numberOfActivities;
		blockActivitiesDuration = (uint8_t*) (blockResourceIndices+cudaData.numberOfResources+1);
		if (cudaData.copySuccessorsMatrixToSharedMemory)
			blockSuccessorsMatrix = blockActivitiesDuration+cudaData.numberOfActivities;
		else
			blockSuccessorsMatrix = cudaData.successorsMatrix;
	}
	__syncthreads();

	for (uint32_t i = threadIdx.x; i < cudaData.numberOfResources+1; i += blockDim.x)	{
		blockResourceIndices[i] = cudaData.resourceIndices[i];
	}

	for (uint32_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)	{
		blockActivitiesDuration[i] = cudaData.activitiesDuration[i];
	}

	if (cudaData.copySuccessorsMatrixToSharedMemory)	{
		for (uint32_t i = threadIdx.x; i < cudaData.successorsMatrixSize; i += blockDim.x)
			blockSuccessorsMatrix[i] = cudaData.successorsMatrix[i];
	}

	// Block have to obtain initial read access.
	if (threadIdx.x == 0)	{
		while (atomicCAS(cudaData.lockSetSolution, DATA_AVAILABLE, DATA_ACCESS) != DATA_AVAILABLE)
			;
		blockBestCost = cudaData.solutionsSetInfo[blockIndexOfSetSolution].solutionCost;
	}
	__syncthreads();

	// Copy solution from a set of solutions to local block order.
	for (uint32_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)	{
		blockCurrentOrder[i] = cudaData.solutionsSet[blockIndexOfSetSolution*cudaData.numberOfActivities+i];
	}
	__syncthreads();

	// Free read lock.
	if (threadIdx.x == 0)	{
		atomicExch(cudaData.lockSetSolution, DATA_AVAILABLE);
	}

	// The best schedule is the current read schedule. It could not be required if data are manipulated in a correct way (no race conditions, no initialization bugs).
	for (uint32_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)	{
		blockBestSolution[i] = blockCurrentOrder[i];
	}

	while (iter < cudaData.numberOfIterationsPerBlock && !blockCriticalPathLengthAchieved)	{

		for (uint16_t i = threadIdx.x+1; i < (cudaData.numberOfActivities-1); i += blockDim.x)	{
			bool relationsBroken = false;
			struct MoveIndices info;
			for (uint16_t j = i+1; j < i+1+cudaData.swapRange; ++j)	{
				info.i = info.j = 0;
				if ((i < cudaData.numberOfActivities-2) && (j < cudaData.numberOfActivities-1) && !relationsBroken)	{
					if (cudaGetMatrixBit(blockSuccessorsMatrix, cudaData.numberOfActivities, blockCurrentOrder[i], blockCurrentOrder[j]) == false)	{
						info.i = i; info.j = j;
					}	else	{
						relationsBroken = true;
					}
				}
				blockReorderingArray[(i-1)*cudaData.swapRange+(j-1-i)] = info;
			}
		}
		__syncthreads();

		uint32_t swapMoves = 0;
		if (blockPartitionCounterUInt16 != NULL)
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArray, (uint32_t*) blockReorderingArrayHelp,  blockPartitionCounterUInt16, maximalNeighbourhoodSize);
		else
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArray, (uint32_t*) blockReorderingArrayHelp,  blockPartitionCounterUInt32, maximalNeighbourhoodSize);

		for (uint32_t i = threadIdx.x; i < swapMoves; i += blockDim.x)	{
			struct MoveIndices *move = &blockReorderingArrayHelp[i];
			if (cudaCheckSwapPrecedencePenalty(blockCurrentOrder, blockSuccessorsMatrix, cudaData.numberOfActivities, move->i, move->j, true) == false)	{
				move->i = move->j = 0;
			}
		}
		__syncthreads();
		
		if (blockPartitionCounterUInt16 != NULL)
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArrayHelp, (uint32_t*) blockReorderingArray,  blockPartitionCounterUInt16, swapMoves);
		else
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArrayHelp, (uint32_t*) blockReorderingArray,  blockPartitionCounterUInt32, swapMoves);

		blockMergeArray[threadIdx.x].cost = 0xffffffff;
		for (uint32_t i = threadIdx.x; i < swapMoves; i += blockDim.x)	{
			struct MoveIndices *move = &blockReorderingArray[i];
			uint32_t threadBestCost = blockMergeArray[threadIdx.x].cost;
			uint32_t totalEval = cudaEvaluateOrder(cudaData, blockCurrentOrder, move->i, move->j,
					blockActivitiesDuration, blockResourceIndices,threadResourcesLoad,
					threadStartValues, threadRemainingResourcesCapacity, threadStartTimesById, cudaData.capacityResolutionAlgorithm);
			totalEval += cudaComputeTardinessPenalty(cudaData.numberOfActivities, blockActivitiesDuration, blockBestCost-1, threadStartTimesById);
			if (totalEval > 0x0000ffff)
				totalEval = 0x0000ffff;
			totalEval <<= 16;
			totalEval |= (hiprand(&threadRandState) & 0x0000ffff);
			uint32_t hashPenalty = 0;
			if (cudaData.useTabuHash == true)	{
				uint32_t hashIdx = cudaComputeHashTableIndex(cudaData.numberOfActivities, blockCurrentOrder, move->i, move->j, move->i, move->j);
				hashPenalty += cudaData.hashMap[hashIdx];
				hashPenalty <<= 16;
			}
			bool isPossibleMove = cudaIsPossibleMove(cudaData.numberOfActivities, move->i, move->j, blockTabuCache);
			if ((isPossibleMove && totalEval+hashPenalty < threadBestCost) || (totalEval>>16) < blockBestCost)	{
				struct MoveInfo newBestThreadSolution = { move->i, move->j, totalEval };
				blockMergeArray[threadIdx.x] = newBestThreadSolution;
			}
		}
		__syncthreads();

		for (uint16_t k = blockDim.x/2; k > 0; k >>= 1)	{
			if (threadIdx.x < k)	{
				if (blockMergeArray[threadIdx.x].cost > blockMergeArray[threadIdx.x+k].cost)
					blockMergeArray[threadIdx.x] = blockMergeArray[threadIdx.x+k];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0)	{
			blockReadPossible = false;
			iterBestMove = blockMergeArray[0];
			iterBestMove.cost >>= 16;
			atomicAdd((unsigned long long*) cudaData.evaluatedSchedules, swapMoves);
			if (iterBestMove.cost < blockBestCost)	{
				blockWriteBestBlock = true;
				blockBestCost = iterBestMove.cost;
				blockNumberOfIterationsSinceBest = 0;
			}

			uint32_t readPossitionCost;
			if (blockReadFromSet == true)
				readPossitionCost = cudaData.solutionsSetInfo[blockIndexOfSetSolution].solutionCost;
			else
				readPossitionCost = *cudaData.globalBestSolutionCost;

			if (blockNumberOfIterationsSinceBest >= blockMaximalNumberOfIterationsSinceBest || readPossitionCost != blockBestCost || *cudaData.globalBestSolutionCost == cudaData.criticalPathLength) {
				bool globalAccess = false, setAccess = false;
				if (atomicCAS(cudaData.lockGlobalSolution, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					globalAccess = true;
				if (atomicCAS(cudaData.lockSetSolution, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					setAccess = true;

				if (globalAccess && setAccess)	{

					if (blockReadFromSet && blockBestCost < cudaData.solutionsSetInfo[blockIndexOfSetSolution].solutionCost)	{
						blockWriteSetSolution = true;
						cudaData.solutionsSetInfo[blockIndexOfSetSolution].readCounter = 0;
						cudaData.solutionsSetInfo[blockIndexOfSetSolution].solutionCost = blockBestCost;
					}	else	{
						atomicExch(cudaData.lockSetSolution, DATA_AVAILABLE);
					}

					if (blockBestCost < *cudaData.globalBestSolutionCost)	{
						blockWriteGlobalBestSolution = true;
						*cudaData.globalBestSolutionCost = blockBestCost;
					}	else	{
						atomicExch(cudaData.lockGlobalSolution, DATA_AVAILABLE);
					}

					if (*cudaData.globalBestSolutionCost == cudaData.criticalPathLength)	{
						blockCriticalPathLengthAchieved = true;
					}

					if (readPossitionCost < blockBestCost || blockNumberOfIterationsSinceBest >= blockMaximalNumberOfIterationsSinceBest)	{
						if (!blockReadSetSolution && !blockReadGlobalBestSolution)	{
							if (blockReadFromSet == true)
								blockReadGlobalBestSolution = true;
							else
								blockReadSetSolution = true;
						}
					}
				} else {
					if (setAccess)
						atomicExch(cudaData.lockSetSolution, DATA_AVAILABLE);
					if (globalAccess)
						atomicExch(cudaData.lockGlobalSolution, DATA_AVAILABLE);
				}
			}  else if (!blockWriteBestBlock)	{
				++blockNumberOfIterationsSinceBest;
			}
		}
		
		if (blockMergeArray[0].cost == 0xffffffff)	{
			// Empty expanded neighborhood. Tabu list will be pruned.
			cudaClearTabuList(cudaData.numberOfActivities, blockTabuList, blockTabuCache, blockTabuListSize/3);
		} else if (threadIdx.x == 0)	{
			// Apply best move.
			uint16_t t = blockCurrentOrder[iterBestMove.i];
			blockCurrentOrder[iterBestMove.i] = blockCurrentOrder[iterBestMove.j];
			blockCurrentOrder[iterBestMove.j] = t;
			// Add move to tabu list.
			cudaAddTurnToTabuList(cudaData.numberOfActivities, iterBestMove.i, iterBestMove.j, blockTabuList, blockTabuCache, blockTabuIdx, blockTabuListSize);
			if (cudaData.useTabuHash == true)	{
				// Add move to hash table.
				uint32_t hashIdx = cudaComputeHashTableIndex(cudaData.numberOfActivities, blockCurrentOrder, 0, 0, iterBestMove.i, iterBestMove.j);
				atomicInc(&cudaData.hashMap[hashIdx], 0xffffffff);
			}
		}
		__syncthreads();

		if (blockWriteBestBlock == true)	{
			if (threadIdx.x == 0)	{
				uint16_t improvedCost = cudaShakingDownEvaluation(cudaData, blockCurrentOrder, threadStartTimesById,
						blockActivitiesDuration, blockResourceIndices, threadResourcesLoad,
						threadStartValues, threadRemainingResourcesCapacity);
				if (improvedCost < blockBestCost)	{
					blockBestCost = improvedCost;
					cudaConvertStartTimesById2ActivitiesOrder(blockCurrentOrder,
							threadStartTimesById, cudaData.numberOfActivities);
				}
			}
			__syncthreads();
			for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
				blockBestSolution[i] = blockCurrentOrder[i];
			blockWriteBestBlock = false;
		}
		__syncthreads();

		if (blockWriteGlobalBestSolution == true)	{
			for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
				cudaData.globalBestSolution[i] = blockBestSolution[i];
			// Copy tabu list + zeros padding.
			for (uint16_t i = threadIdx.x; i < cudaData.maxTabuListSize; i += blockDim.x)
				cudaData.globalBestSolutionTabuList[i] = blockTabuList[i];
			__threadfence();
			__syncthreads();
			if (threadIdx.x == 0)	{
				blockWriteGlobalBestSolution = false;
				atomicExch(cudaData.lockGlobalSolution, DATA_AVAILABLE);
			}
		}

		if (blockWriteSetSolution == true)	{
			for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
				cudaData.solutionsSet[blockIndexOfSetSolution*cudaData.numberOfActivities+i] = blockBestSolution[i];
			for (uint16_t i = threadIdx.x; i < cudaData.maxTabuListSize; i += blockDim.x)
				cudaData.solutionSetTabuLists[blockIndexOfSetSolution*cudaData.maxTabuListSize+i] = blockTabuList[i];
			__threadfence();
			__syncthreads();
			if (threadIdx.x == 0)	{
				blockWriteSetSolution = false;
				atomicExch(cudaData.lockSetSolution, DATA_AVAILABLE);
			}
		}

		if (blockReadGlobalBestSolution == true)	{
			if (threadIdx.x == 0)	{
				if (atomicCAS(cudaData.lockGlobalSolution, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					blockReadPossible = true;
			}
			__syncthreads();
			if (blockReadPossible)	{
				// Read global best solution to memory.
				cudaReadExternalSolution(cudaData.numberOfActivities, blockTabuList, blockTabuCache, blockTabuListSize,
						blockCurrentOrder, cudaData.globalBestSolution, cudaData.globalBestSolutionTabuList);
				if (threadIdx.x == 0)	{
					blockBestCost = *cudaData.globalBestSolutionCost;
					blockNumberOfIterationsSinceBest = 0;

					blockReadFromSet = false;
					blockReadGlobalBestSolution = false;
					blockMaximalNumberOfIterationsSinceBest = hiprand(&randState) % cudaData.maximalIterationsSinceBest;
					atomicExch(cudaData.lockGlobalSolution, DATA_AVAILABLE);
				}
			}
		}

		if (blockReadSetSolution == true)	{
			if (threadIdx.x == 0)	{
				if (atomicCAS(cudaData.lockSetSolution, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					blockReadPossible = true;
			}
			__syncthreads();
			if (blockReadPossible)	{
				if (threadIdx.x == 0)	{
					blockIndexOfSetSolution = (blockIndexOfSetSolution+1) % cudaData.solutionsSetSize;
				}
				__syncthreads();
				// Read solution from a set to block memory.
				cudaReadExternalSolution(cudaData.numberOfActivities, blockTabuList, blockTabuCache, blockTabuListSize, blockCurrentOrder,
						cudaData.solutionsSet+blockIndexOfSetSolution*cudaData.numberOfActivities, cudaData.solutionSetTabuLists+blockIndexOfSetSolution*cudaData.maxTabuListSize);
				if (threadIdx.x == 0)	{
					blockBestCost = cudaData.solutionsSetInfo[blockIndexOfSetSolution].solutionCost;
					uint32_t readCounter = ++cudaData.solutionsSetInfo[blockIndexOfSetSolution].readCounter;
					blockNumberOfIterationsSinceBest = 0;

					blockReadFromSet = true;
					blockReadSetSolution = false;
					blockMaximalNumberOfIterationsSinceBest = hiprand(&randState) % cudaData.maximalIterationsSinceBest;
					atomicExch(cudaData.lockSetSolution, DATA_AVAILABLE);
					if (readCounter > cudaData.maximalValueOfReadCounter)
						cudaDiversificationOfSolution(cudaData.numberOfActivities, blockCurrentOrder, blockSuccessorsMatrix, cudaData.numberOfDiversificationSwaps, &randState);
				}
			}
		}

		if (threadIdx.x == 0)	{
			++iter;
		}
		__syncthreads();
	}

	// Write solution if is better than best found.
	if (threadIdx.x == 0)	{
		while (atomicCAS(cudaData.lockGlobalSolution, DATA_AVAILABLE, DATA_ACCESS) != DATA_AVAILABLE)
			;
	}
	__syncthreads();

	if (*cudaData.globalBestSolutionCost > blockBestCost)	{
		for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
			cudaData.globalBestSolution[i] = blockBestSolution[i];
		for (uint16_t i = threadIdx.x; i < cudaData.maxTabuListSize; i += blockDim.x)
			cudaData.globalBestSolutionTabuList[i] = blockTabuList[i];
		if (threadIdx.x == 0)
			*cudaData.globalBestSolutionCost = blockBestCost;
	}
	__syncthreads();

	if (threadIdx.x == 0)
		atomicExch(cudaData.lockGlobalSolution, DATA_AVAILABLE);

	return;
}


/* START MAIN CUDA KERNEL */

void runCudaSolveRCPSP(int numberOfBlock, int numberOfThreadsPerBlock, int computeCapability, int dynSharedMemSize, const CudaData& cudaData)	{
	if (dynSharedMemSize < 7950)	{
		// Prefare 16 kB shared memory + 48 kB cache L1.
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaSolveRCPSP), hipFuncCachePreferL1);
	} else {
		// Prefare 48 kB shared memory + 16 kB cache L1.
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaSolveRCPSP), hipFuncCachePreferShared);
	}
	// Set maximum amount of dynamic memory to 1 MB.
	hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024);
	// Launch the main GPU kernel.
	cudaSolveRCPSP<<<numberOfBlock,numberOfThreadsPerBlock,dynSharedMemSize>>>(cudaData);
	hipDeviceSynchronize();
}

