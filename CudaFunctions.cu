#include "hip/hip_runtime.h"
/*!
 * \file CudaFunctions.cu
 * \author Libor Bukata
 * \brief RCPSP Cuda functions.
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "CudaConstants.h"
#include "CudaFunctions.cuh"

#if defined _WIN32 || defined _WIN64 || defined WIN32 || defined WIN64
#include "hip/hip_runtime.h"
#include ""
#endif

using std::cerr;
using std::cout;
using std::endl;

//! Texture reference of activities resource requirements.
texture<uint8_t,1,hipReadModeElementType> cudaActivitiesResourcesTex;
//! Texture reference of predecessors.
texture<uint16_t,1,hipReadModeElementType> cudaPredecessorsTex;
//! Texture reference of predecessors indices.
texture<uint16_t,1,hipReadModeElementType> cudaPredecessorsIndicesTex;

//! The longest paths from the end dummy activity to the others in the transformed graph.
__constant__ uint16_t rightLeftLongestPaths[NUMBER_OF_ACTIVITIES];


/* CUDA BIND TEXTURES */

int bindTexture(void *texData, int32_t arrayLength, int option)	{
	switch (option)	{
		case ACTIVITIES_RESOURCES:
			return hipBindTexture(NULL, cudaActivitiesResourcesTex, texData, arrayLength*sizeof(uint8_t));
		case PREDECESSORS:
			return hipBindTexture(NULL, cudaPredecessorsTex, texData, arrayLength*sizeof(uint16_t));
		case PREDECESSORS_INDICES:
			return hipBindTexture(NULL, cudaPredecessorsIndicesTex, texData, arrayLength*sizeof(uint16_t));
		default:
			cerr<<"bindTextures: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}

int unbindTexture(int option)	{
	switch (option)	{
		case ACTIVITIES_RESOURCES:
			return hipUnbindTexture(cudaActivitiesResourcesTex);
		case PREDECESSORS:
			return hipUnbindTexture(cudaPredecessorsTex);
		case PREDECESSORS_INDICES:
			return hipUnbindTexture(cudaPredecessorsIndicesTex);
		default:
			cerr<<"unbindTextures: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}

int memcpyToSymbol(void *source, int32_t arrayLength, int option)	{
	switch (option)	{
		case THE_LONGEST_PATHS:
			return hipMemcpyToSymbol(HIP_SYMBOL(rightLeftLongestPaths), (void*) source, arrayLength*sizeof(uint16_t));
		default:
			cerr<<"memcpyToSymbol: Invalid option!"<<endl;
	}
	return hipErrorInvalidValue;
}


/* CUDA IMPLEMENT OF SOURCES LOAD - CAPACITY RESOLUTION  */

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \brief Prepare arrays for next use (schedule evaluation).
 */
inline __device__ void cudaPrepareArrays(const CudaData& cudaData, uint16_t *& resourcesLoad, uint16_t *& startValues)	{
	for (uint16_t i = 0; i < cudaData.sumOfCapacities; ++i)
		resourcesLoad[i] = 0;
	for (uint16_t i = 0; i < cudaData.maximalCapacityOfResource; ++i)
		startValues[i] = 0;
}

/*!
 * \param numberOfResources Number of resources.
 * \param activityId Activity identification.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param resourceIndices Access indices for resources.
 * \return Earliest start time of an activity.
 * \brief Function return earliest possible start time of an activity. Precedence relations are ignored.
 */
inline __device__ uint16_t cudaGetEarliestStartTime(const uint16_t& numberOfResources, const uint16_t& activityId, uint16_t *&resourcesLoad, uint16_t *&resourceIndices) {
	uint16_t bestStart = 0;
	for (uint8_t resourceId = 0; resourceId < numberOfResources; ++resourceId)	{
		uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		if (activityRequirement > 0)
			bestStart = max(resourcesLoad[resourceIndices[resourceId+1]-activityRequirement], bestStart);
	}
	return bestStart;
}

/*!
 * \param activityId Activity identification.
 * \param activityStart Start time of an activity.
 * \param activityStop Stop time of an activity.
 * \param numberOfResources Number of resources.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \brief Function add new activity and update resources arrays. Irreversible process.
 */
inline __device__ void cudaAddActivity(const uint16_t& activityId, const uint16_t& activityStart, const uint16_t& activityStop,
		const uint16_t& numberOfResources, uint16_t *&resourceIndices,  uint16_t *&resourcesLoad, uint16_t *&startValues)	{
	int32_t requiredSquares, timeDiff;
	int32_t c, k, capacityOfResource, resourceRequirement, newStartTime, resourceStartIdx;
	for (uint8_t resourceId = 0; resourceId < numberOfResources; ++resourceId)	{
		resourceStartIdx = resourceIndices[resourceId];
		capacityOfResource = resourceIndices[resourceId+1]-resourceStartIdx;
		resourceRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		requiredSquares = resourceRequirement*(activityStop-activityStart);
		if (requiredSquares > 0)	{
			c = 0; k = 0;
			newStartTime = activityStop;
			while (requiredSquares > 0 && k < capacityOfResource)	{
				if (resourcesLoad[resourceStartIdx+k] < newStartTime)	{
					if (c >= resourceRequirement)
						newStartTime = startValues[c-resourceRequirement];
					timeDiff = newStartTime-max(resourcesLoad[resourceStartIdx+k], activityStart);
					if (requiredSquares-timeDiff > 0)	{
						requiredSquares -= timeDiff;
						startValues[c++] = resourcesLoad[resourceStartIdx+k];
						resourcesLoad[resourceStartIdx+k] = newStartTime;
					} else {
						resourcesLoad[resourceStartIdx+k] = newStartTime-timeDiff+requiredSquares;
						break;
					}
				}
				++k;
			}
		}
	}
}

/* CUDA IMPLEMENT OF SOURCES LOAD - TIME RESOLUTION  */

/*!
 * \param numberOfActivities Number of activities in the project.
 * \param numberOfResources Number of renewable resources in the project.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param resourceIndices Access indices for resources.
 * \brief It initializes vectors of free capacities to initial values (capacities of resources).
 */
inline __device__ void cudaPrepareArrays(const uint16_t& numberOfActivities, const uint16_t& numberOfResources, const uint32_t& UBTime,
	       	uint8_t *& remainingResourcesCapacity, uint16_t *& resourceIndices)	{
	for (uint16_t resourceId = 0; resourceId < numberOfResources; ++resourceId)
		for (uint32_t t = 0; t < UBTime; ++t)
			remainingResourcesCapacity[resourceId*UBTime+t] = resourceIndices[resourceId+1]-resourceIndices[resourceId];
}

/*!
 * \param numberOfResources Number of renewable resources in the project.
 * \param activityId Identification of the activity that should be added (required for texture memory access).
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param precTime The earliest precedence violation free start time of the activity activityId.
 * \param activityDuration Duration of the activity activityId.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \return The earliest start time of the activity without resource overload.
 * \brief It finds out the earliest start time of the activity activityId.
 */
inline __device__ uint16_t cudaGetEarliestStartTime(const uint16_t& numberOfResources, const uint16_t& activityId,
		uint8_t *&remainingResourcesCapacity, const uint16_t& precTime, int32_t activityDuration, const uint32_t& UBTime) {
	int32_t loadTime = 0, t = UBTime;
	for (t = precTime; t < UBTime && loadTime < activityDuration; ++t)       {
		bool capacityAvailable = true;
		for (int32_t resourceId = 0; resourceId < numberOfResources && capacityAvailable; ++resourceId)        {
			uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
			if (remainingResourcesCapacity[resourceId*UBTime+t] < activityRequirement)	{
				loadTime = 0;
				capacityAvailable = false;
			}
		}
		if (capacityAvailable == true)
			++loadTime;
	}
	return (uint16_t) t-loadTime;
}

/*!
 * \param activityId Identification of the added activity.
 * \param activityStart Scheduled start time of the activity.
 * \param activityStop Scheduled finish time of the activity.
 * \param numberOfResources Number of renewable resources in the project.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param UBTime Upper bound of the maximal duration of the project.
 * \brief It updates the state of all resources after activity is added.
 */
inline __device__ void cudaAddActivity(const uint16_t& activityId, const uint16_t& activityStart, const uint16_t& activityStop,
		const uint16_t& numberOfResources, uint8_t *&remainingResourcesCapacity, const uint32_t& UBTime)	{
	for (int32_t resourceId = 0; resourceId < numberOfResources; ++resourceId)     {
		uint8_t activityRequirement = tex1Dfetch(cudaActivitiesResourcesTex, activityId*numberOfResources+resourceId);
		for (uint32_t t = activityStart; t < activityStop; ++t)
			remainingResourcesCapacity[resourceId*UBTime+t] -= activityRequirement;
	}
}

/* CUDA IMPLEMENTATION OF THE BASE RESOURCE EVALUATION FUNCTIONS */

/*!
 * \param cudaData RCPSP constants, variables, ...
 * \param blockOrder Current order of the activities.
 * \param indexI Swap index i.
 * \param indexJ Swap index j.
 * \param activitiesDuration Duration of the activities.
 * \param resourceIndices Access indices for resources.
 * \param resourcesLoad Array of the earliest resource start times.
 * \param startValues Helper array for resource evaluation.
 * \param remainingResourcesCapacity Free capacity of each resource with respect to time.
 * \param startTimesWriterById Array of start times of the scheduled activities ordered by ID's.
 * \param capacityResolution If true then capacity based algorithm is selected else time based algorithm is selected.
 * \return Schedule length without any penalties.
 * \brief Function evaluate schedule and return total schedule length.
 */
inline __device__ uint16_t cudaEvaluateOrder(const CudaData& cudaData, uint16_t *&blockOrder, const uint16_t& indexI, const uint16_t& indexJ, uint8_t *&activitiesDuration, uint16_t *&resourceIndices,
		uint16_t *resourcesLoad, uint16_t *startValues, uint8_t *remainingResourcesCapacity, uint16_t *startTimesWriterById, bool capacityResolution)	{

	// Current cost of the schedule.
	uint16_t scheduleLength = 0;

	// Init state of resources.
	if (capacityResolution == true)
		cudaPrepareArrays(cudaData, resourcesLoad, startValues);
	else
		cudaPrepareArrays(cudaData.numberOfActivities, cudaData.numberOfResources, MAXIMAL_SUM_OF_FLOATS, remainingResourcesCapacity, resourceIndices);
	
	for (uint16_t i = 0; i < cudaData.numberOfActivities; ++i)	{

		uint16_t activityId = blockOrder[i];

		// Logical swap.
		if (i == indexI)
			activityId = blockOrder[indexJ];

		if (i == indexJ)
			activityId = blockOrder[indexI];

		// Get the earliest start time without precedence penalty. (if moves are precedence penalty free)
		uint16_t start = 0;
		uint32_t baseIndex = tex1Dfetch(cudaPredecessorsIndicesTex, activityId);
		uint16_t numberOfPredecessors = tex1Dfetch(cudaPredecessorsIndicesTex, activityId+1)-baseIndex;
		for (uint16_t j = 0; j < numberOfPredecessors; ++j)	{
			uint16_t predecessorId = tex1Dfetch(cudaPredecessorsTex, baseIndex+j);
			start = max(startTimesWriterById[predecessorId]+activitiesDuration[predecessorId], start);
		}

		// Get the earliest start time if the resources restrictions are counted.
		if (capacityResolution == true)
			start = max(cudaGetEarliestStartTime(cudaData.numberOfResources, activityId, resourcesLoad, resourceIndices), start);
		else
			start = max(cudaGetEarliestStartTime(cudaData.numberOfResources, activityId, remainingResourcesCapacity, start, activitiesDuration[activityId], MAXIMAL_SUM_OF_FLOATS), start);

		// Add activity = update resources arrays + write start time.
		uint16_t stop = start+activitiesDuration[activityId];
		if (capacityResolution == true)
			cudaAddActivity(activityId, start, stop, cudaData.numberOfResources, resourceIndices, resourcesLoad, startValues);
		else
			cudaAddActivity(activityId, start, stop, cudaData.numberOfResources, remainingResourcesCapacity, MAXIMAL_SUM_OF_FLOATS);

		startTimesWriterById[activityId] = start;
		scheduleLength = max(scheduleLength, stop);
	}

	return scheduleLength;
}


/* CHECK PRECEDENCE FUNCTIONS */

/*!
 * \param successorsMatrix Bit matrix of successors.
 * \param numberOfActivities Number of activities.
 * \param activityId1 Activity identification.
 * \param activityId2 Activity identification.
 * \return True if an activity with identification activityId2 is successor of an activity with identification activityId1.
 * \brief Check if activity ID2 is successor of activity ID1.
 */
inline __device__ bool cudaGetMatrixBit(const uint8_t * const& successorsMatrix, const uint16_t& numberOfActivities, const int16_t& activityId1, const int16_t& activityId2)	{
	uint32_t bitPossition = activityId1*numberOfActivities+activityId2;
	if ((successorsMatrix[bitPossition/8] & (1<<(bitPossition % 8))) > 0)
		return true;
	else
		return false;
}

/*!
 * \param data constants, variables and data.
 * \param order Sequence of activities.
 * \param successorsMatrix Bit matrix of successors.
 * \param i Index i of swap.
 * \param j Index j of swap.
 * \param light If true then light version is executed. (precedences from activity at index i aren't checked)
 * \return True if current swap won't break relation precedences else false.
 * \brief Check if requested move is precedence penalty free.
 */
inline __device__ bool cudaCheckSwapPrecedencePenalty(const CudaData& data, const uint16_t * const& order, const uint8_t * const& successorsMatrix, int16_t i, int16_t j, bool light = false)	{
	if (i > j)	{
		int16_t t = i;
		i = j; j = t;
	}
	for (uint16_t k = i; k < j; ++k)	{
		if (cudaGetMatrixBit(successorsMatrix, data.numberOfActivities, order[k], order[j]) == true)
			return false;
	}
	if (!light)	{
		for (uint16_t k = i+1; k < j; ++k)	{
			if (cudaGetMatrixBit(successorsMatrix, data.numberOfActivities, order[i], order[k]) == true)
				return false;
		}
	}
	return true;
}

/*!
 * \param numAct The number of activities.
 * \param successorsMatrix Binary matrix of successors.
 * \param activitiesDuration Duration of each activity.
 * \param startTimesById Array of start time values of the scheduled activities ordered by ID's.
 * \return The precedence penalty.
 * \brief It finds out all precedence penalties and computes penalty.
 * \note The penalty should be zero since only non-precedence breaking moves are allowed.
 */
__device__ uint32_t cudaComputePrecedencePenalty(uint16_t numAct, uint8_t *successorsMatrix, uint8_t *activitiesDuration, uint16_t *startTimesById)  {
	uint32_t penalty = 0;
	for (uint16_t id1 = 0; id1 < numAct; ++id1)        {
		for (uint16_t id2 = 0; id2 < numAct; ++id2)        {
			if (id1 != id2 && cudaGetMatrixBit(successorsMatrix, numAct, id1, id2) == true)	{
				if (startTimesById[id1]+activitiesDuration[id1] > startTimesById[id2])
					penalty += startTimesById[id1]+activitiesDuration[id1]-startTimesById[id2];
			}

		}
	}
	return penalty;
}

/* SOFT VIOLATION PENALTIES */

/*!
 * \param numberOfActivities The number of the activities in the project.
 * \param activitiesDuration Duration of each activity.
 * \param makespan The best known project makespan.
 * \param startTimesById Array of start time values of the scheduled activities ordered by ID's.
 * \return It returns overall tardiness penalty.
 */
inline __device__ uint32_t cudaComputeTardinessPenalty(uint16_t numberOfActivities, uint8_t *activitiesDuration, uint32_t makespan, uint16_t *startTimesById)	{
	uint32_t overhangPenalty = 0;
	for (uint16_t id = 0; id < numberOfActivities; ++id)	{
		if (startTimesById[id]+activitiesDuration[id]+rightLeftLongestPaths[id] > makespan)
			overhangPenalty += startTimesById[id]+activitiesDuration[id]+rightLeftLongestPaths[id]-makespan;
	}
	return overhangPenalty;
}

/*	CUDA IMPLEMENT OF SIMPLE TABU LIST */

/*!
 * \param numberOfActivities Number of activities.
 * \param i Swap index i.
 * \param j Swap index j.
 * \param tabuCache Block tabu cache - fast check if move is in tabu list.
 * \return True if move is possible else false.
 * \brief Check if move is in tabu list.
 */
inline __device__ bool cudaIsPossibleMove(const uint16_t& numberOfActivities, const uint16_t& i, const uint16_t& j, uint8_t *&tabuCache)	{
	if (tabuCache[i*numberOfActivities+j] == 0 || tabuCache[j*numberOfActivities+i] == 0)
		return true;
	else
		return false;
}

/*!
 * \param numberOfActivities Number of activities.
 * \param i Swap index i of added move.
 * \param j Swap index j of added move.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param tabuIdx Current index at tabu list.
 * \param tabuListSize Tabu list size.
 * \brief Add specified move to tabu list and update tabu cache.
 */
inline __device__ void cudaAddTurnToTabuList(const uint16_t& numberOfActivities, const uint16_t& i, const uint16_t& j, MoveIndices *&tabuList, uint8_t *&tabuCache, uint16_t& tabuIdx, const uint16_t& tabuListSize)	{

	MoveIndices move = tabuList[tabuIdx];
	uint16_t iOld = move.i, jOld = move.j;

	if (iOld != 0 && jOld != 0)
		tabuCache[iOld*numberOfActivities+jOld] = tabuCache[jOld*numberOfActivities+iOld] = 0;

	move.i = i; move.j = j;
	tabuList[tabuIdx] = move;
	tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 1;

	tabuIdx = (tabuIdx+1) % tabuListSize;
}


/* HELP FUNCTIONS */

/*!
 * \param numberOfActivities Number of activities.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param numberOfElements Number of tabu list elements that will be removed.
 * \brief Remove specified number of elements from tabu list and update tabu cache.
 */
inline __device__ void cudaClearTabuList(const uint16_t& numberOfActivities, MoveIndices *tabuList, uint8_t *tabuCache, const uint16_t& numberOfElements)	{
	for (uint16_t k = threadIdx.x; k < numberOfElements; k += blockDim.x)	{
		MoveIndices *tabuMove = &tabuList[k];
		uint16_t i = tabuMove->i, j = tabuMove->j;
		tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 0;
		tabuMove->j = tabuMove->i = 0;
	}
	__syncthreads();
	return;
}

/*!
 * \param numberOfActivities Number of activities.
 * \param tabuList Tabu list.
 * \param tabuCache Tabu cache.
 * \param tabuListSize Block tabu list size.
 * \param blockOrder Block schedule - order.
 * \param externalSolution Solution from a set or the best global solution. (order)
 * \param externalTabuList Tabu list of external solution.
 * \brief Replace current block solution with a read external solution (order+tabu).
 */
inline __device__ void cudaReadExternalSolution(const uint16_t& numberOfActivities, MoveIndices *tabuList, uint8_t *tabuCache, const uint16_t& tabuListSize,
		uint16_t *blockOrder, uint16_t *externalSolution, MoveIndices *externalTabuList)	{
	// Clear current tabu list and tabu cache.
	cudaClearTabuList(numberOfActivities, tabuList, tabuCache, tabuListSize);
	// Read block order.
	for (uint16_t i = threadIdx.x; i < numberOfActivities; i += blockDim.x)
		blockOrder[i] = externalSolution[i];
	// Read block tabu list and create tabu cache.
	for (uint16_t l = threadIdx.x; l < tabuListSize; l += blockDim.x)	{
		tabuList[l] = externalTabuList[l];
		MoveIndices *move = &tabuList[l];
		uint16_t i = move->i, j = move->j;
		tabuCache[i*numberOfActivities+j] = tabuCache[j*numberOfActivities+i] = 1;
	}
	__syncthreads();
	return;
}

/* REORDER ARRAY FUNCTION */

/*!
 * \tparam T uint16_t or uint32_t.
 * \param moves Array of moves which should be reorder.
 * \param resultMerge Result array of reordered moves.
 * \param threadsCounter Helper array for threads counters.
 * \param size How many elements will be processed at moves array.
 * \return Number of written elements to resultMerge array.
 * \brief Move all valid moves to the resultMerge array and return number of valid moves.
 */
template <typename T>
inline __device__ uint32_t cudaReorderMoves(uint32_t *moves, uint32_t *resultMerge, T *threadsCounter, const uint32_t& size)	{
	threadsCounter[threadIdx.x] = 0;
	uint32_t threadAmount = size/blockDim.x+1;
	for (uint32_t i = threadIdx.x*threadAmount; i < size && i < (threadIdx.x+1)*threadAmount; ++i)	{
		if (moves[i] != 0)
			++threadsCounter[threadIdx.x];
	}
	__syncthreads();
	for (uint32_t k = 0; (1<<k) < blockDim.x; ++k)   {
		uint32_t step = 1<<k;
		uint32_t begIdx = (step-1)+2*step*threadIdx.x;
		if (begIdx < blockDim.x-step)
			threadsCounter[begIdx+step] += threadsCounter[begIdx];
		__syncthreads();
	}
	for (int32_t k = (blockDim.x>>1); k > 1; k >>= 1)	{
		uint32_t step = k/2;
		uint32_t begIdx = (k-1)+2*step*threadIdx.x;
		if (begIdx < blockDim.x-step) 
			threadsCounter[begIdx+step] += threadsCounter[begIdx];
		__syncthreads();
	}
	uint32_t threadStartIndex = threadIdx.x > 0 ? threadsCounter[threadIdx.x-1] : 0;
	for (uint32_t i = threadIdx.x*threadAmount; i < size && i < (threadIdx.x+1)*threadAmount; ++i)	{
		if (moves[i] != 0)
			resultMerge[threadStartIndex++] = moves[i];
	}
	__syncthreads();
	return threadsCounter[blockDim.x-1];
}

/* DIVERSIFICATION FUNCTION */

/*!
 * \param data constants, variables and data.
 * \param order Current schedule - sequence of activities.
 * \param successorsMatrix Bit matrix of successors.
 * \param diversificationSwaps Number of diversification swaps.
 * \param state State of the random generator.
 * \brief Function performs specified number of precedence penalty free swaps.
 */
inline __device__ void cudaDiversificationOfSolution(const CudaData& data, uint16_t *order, const uint8_t *successorsMatrix, const uint32_t& diversificationSwaps, hiprandState *state)	{
	uint32_t performedSwaps = 0;
	while (performedSwaps < diversificationSwaps)  {
		uint16_t i = (hiprand(state) % (data.numberOfActivities-2)) + 1;
		uint16_t j = (hiprand(state) % (data.numberOfActivities-2)) + 1;
		if ((i != j) && (cudaCheckSwapPrecedencePenalty(data, order, successorsMatrix, i, j) == true))	{
			uint16_t t = order[i];
			order[i] = order[j];
			order[j] = t;
			++performedSwaps;
		}
	}
	return;
}

/* HEURISTIC - DIVIDING ITERATIONS AMONG SOLUTIONS */

/*!
 * \param data Constants, variables and pointers to the data-structures.
 * \param indexOfSetSolution The index of the loaded solution.
 * \return The number of assigned iterations to the loaded solution.
 * \brief Iterations Balancing Heuristic is dividing work among solutions according to their quality and the number of iterations already performed on them.
 */
inline __device__ uint32_t calculateTheNumberOfAssignedIterationsSinceLoad(const CudaData& data, const uint32_t& indexOfSetSolution)	{
	uint32_t quantity = (gridDim.x*data.numberOfIterationsPerBlock)/(5*data.totalSolutions);
	float p1 = (((float) data.infoAboutSolutions[indexOfSetSolution].iterationCounter)/((float) data.numberOfIterationsPerBlock));
	float p2 = ((((float) data.infoAboutSolutions[indexOfSetSolution].solutionCost)/((float) *data.bestSolutionCost))-1.0f);
	return (uint32_t) (quantity*(0.8f*expf(-100.0f*p2)+0.2f*expf(-4.0f*p1)));
}


/*	CUDA IMPLEMENT OF GLOBAL KERNEL */

/*!
 * Global function dealing with the RCPSP problem. Blocks communicate with each other through global memory.
 * Local variables are coalesced. Dynamic shared memory, texture memory and constant memory are used.
 * \param cudaData All required constants, pointers to device memory, setting variables, ....
 * \brief Solve the RCPSP problem using GPU.
 */
__global__ void cudaSolveRCPSP(const CudaData cudaData)	{
	
	__shared__ uint32_t iter;
	__shared__ MoveInfo iterBestMove;
	__shared__ uint32_t blockBestCost;
	__shared__ uint16_t *blockBestSolution;
	__shared__ uint32_t maximalNeighbourhoodSize;
	__shared__ uint8_t *blockActivitiesDuration;
	__shared__ uint16_t *blockCurrentOrder;
	__shared__ uint8_t *blockSuccessorsMatrix;
	__shared__ MoveInfo *blockMergeArray;
	__shared__ float blockUniformProbability;
	__shared__ uint16_t *blockPartitionCounterUInt16;
	__shared__ uint32_t *blockPartitionCounterUInt32;
	__shared__ MoveIndices *blockReorderingArray;
	__shared__ MoveIndices *blockReorderingArrayHelp;

	__shared__ uint16_t blockTabuIdx;
	__shared__ uint16_t blockTabuListSize;
	__shared__ MoveIndices *blockTabuList;
	__shared__ uint8_t *blockTabuCache;
	__shared__ int32_t blockIndexOfSetSolution;
	__shared__ bool blockReadPossible;
	__shared__ bool blockWriteBestBlock;
	__shared__ bool blockReadSetSolution;
	__shared__ bool blockWriteSetSolution;
	__shared__ bool blockCriticalPathLengthAchieved;
	__shared__ uint32_t blockIterationsSinceImprovement;
	__shared__ uint32_t blockNumberOfIterationsSinceLoad;
	__shared__ uint32_t blockMaximalNumberOfIterationsSinceLoad;
	__shared__ uint16_t *blockResourceIndices;

	__shared__ hiprandState randState;

	hiprandState threadRandState;
	hiprand_init(blockDim.x*blockIdx.x+threadIdx.x, threadIdx.x, 0, &threadRandState);

	uint16_t threadResourcesLoad[TOTAL_SUM_OF_CAPACITY];
	uint16_t threadStartValues[MAXIMUM_CAPACITY_OF_RESOURCE];
	uint8_t threadRemainingResourcesCapacity[NUMBER_OF_RESOURCES*MAXIMAL_SUM_OF_FLOATS];
	uint16_t threadStartTimesById[NUMBER_OF_ACTIVITIES];

	extern __shared__ uint8_t dynamicSharedMemory[];
	if (threadIdx.x == 0)	{
		/* SET VARIABLES */
		iter = 0;
		blockTabuIdx = 0;
		blockWriteBestBlock = false;
		blockReadSetSolution = false;
		blockWriteSetSolution = false;
		blockCriticalPathLengthAchieved= false;
		blockIterationsSinceImprovement = 0;
		blockNumberOfIterationsSinceLoad = 0;
		blockIndexOfSetSolution = blockIdx.x % cudaData.totalSolutions;
		maximalNeighbourhoodSize = (cudaData.numberOfActivities-2)*cudaData.swapRange;
		blockReorderingArray = cudaData.swapMergeArray+blockIdx.x*maximalNeighbourhoodSize;
		blockReorderingArrayHelp = cudaData.mergeHelpArray+blockIdx.x*maximalNeighbourhoodSize;
		blockTabuList = cudaData.tabuLists+blockIdx.x*cudaData.maxTabuListSize;
		blockTabuListSize = cudaData.maxTabuListSize-((cudaData.maxTabuListSize*blockIdx.x)/(4*gridDim.x));
		blockTabuCache = cudaData.tabuCaches+blockIdx.x*cudaData.numberOfActivities*cudaData.numberOfActivities;
		blockBestSolution = cudaData.blocksBestSolution+blockIdx.x*cudaData.numberOfActivities;

		hiprand_init(3*blockIdx.x+71, blockIdx.x, 0, &randState);


		/* ASSIGN SHARED MEMORY */

		// It is necessary to use an offset to have the aligned memory!
		blockMergeArray = (MoveInfo*) &dynamicSharedMemory[sizeof(MoveInfo)-(((uint64_t) dynamicSharedMemory) % sizeof(MoveInfo))];

		if (maximalNeighbourhoodSize < 0xffff)	{
			blockPartitionCounterUInt16 = (uint16_t*) (blockMergeArray+blockDim.x);
			blockPartitionCounterUInt32 = NULL;
			blockCurrentOrder = blockPartitionCounterUInt16+blockDim.x;
		} else	{
			blockPartitionCounterUInt32 = (uint32_t*) (blockMergeArray+blockDim.x);
			blockPartitionCounterUInt16 = NULL;
			blockCurrentOrder = (uint16_t*) (blockPartitionCounterUInt32+blockDim.x);
		}	
		blockResourceIndices = blockCurrentOrder+cudaData.numberOfActivities;
		blockActivitiesDuration = (uint8_t*) (blockResourceIndices+cudaData.numberOfResources+1);
		if (cudaData.copySuccessorsMatrixToSharedMemory)
			blockSuccessorsMatrix = blockActivitiesDuration+cudaData.numberOfActivities;
		else
			blockSuccessorsMatrix = cudaData.successorsMatrix;
	}
	__syncthreads();

	for (uint32_t i = threadIdx.x; i < cudaData.numberOfResources+1; i += blockDim.x)	{
		blockResourceIndices[i] = cudaData.resourceIndices[i];
	}

	for (uint32_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)	{
		blockActivitiesDuration[i] = cudaData.durationOfActivities[i];
	}

	if (cudaData.copySuccessorsMatrixToSharedMemory)	{
		for (uint32_t i = threadIdx.x; i < cudaData.successorsMatrixSize; i += blockDim.x)
			blockSuccessorsMatrix[i] = cudaData.successorsMatrix[i];
	}

	// Block have to obtain initial read access.
	if (threadIdx.x == 0)	{
		while (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) != DATA_AVAILABLE)
			;
		blockBestCost = cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost;
	}
	__syncthreads();

	// Copy solution from a set of solutions to local block order.
	for (uint32_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)	{
		blockCurrentOrder[i] = cudaData.ordersOfSolutions[blockIndexOfSetSolution*cudaData.numberOfActivities+i];
	}
	__syncthreads();

	// Calculate assigned number of iterations and free read-only lock.
	if (threadIdx.x == 0)	{
		blockMaximalNumberOfIterationsSinceLoad = calculateTheNumberOfAssignedIterationsSinceLoad(cudaData, blockIndexOfSetSolution);
		atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
	}

	while (iter < cudaData.numberOfIterationsPerBlock && !blockCriticalPathLengthAchieved)	{

		for (uint16_t i = threadIdx.x+1; i < (cudaData.numberOfActivities-1); i += blockDim.x)	{
			bool relationsBroken = false;
			struct MoveIndices info;
			for (uint16_t j = i+1; j < i+1+cudaData.swapRange; ++j)	{
				info.i = info.j = 0;
				if ((i < cudaData.numberOfActivities-2) && (j < cudaData.numberOfActivities-1) && !relationsBroken)	{
					if (cudaGetMatrixBit(blockSuccessorsMatrix, cudaData.numberOfActivities, blockCurrentOrder[i], blockCurrentOrder[j]) == false)	{
						info.i = i; info.j = j;
					}	else	{
						relationsBroken = true;
					}
				}
				blockReorderingArray[(i-1)*cudaData.swapRange+(j-1-i)] = info;
			}
		}
		__syncthreads();

		uint32_t swapMoves = 0;
		if (blockPartitionCounterUInt16 != NULL)
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArray, (uint32_t*) blockReorderingArrayHelp,  blockPartitionCounterUInt16, maximalNeighbourhoodSize);
		else
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArray, (uint32_t*) blockReorderingArrayHelp,  blockPartitionCounterUInt32, maximalNeighbourhoodSize);

		for (uint32_t i = threadIdx.x; i < swapMoves; i += blockDim.x)	{
			struct MoveIndices *move = &blockReorderingArrayHelp[i];
			if (cudaCheckSwapPrecedencePenalty(cudaData, blockCurrentOrder, blockSuccessorsMatrix, move->i, move->j, true) == false)	{
				move->i = move->j = 0;
			}
		}
		__syncthreads();
		
		if (blockPartitionCounterUInt16 != NULL)
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArrayHelp, (uint32_t*) blockReorderingArray,  blockPartitionCounterUInt16, swapMoves);
		else
			swapMoves = cudaReorderMoves((uint32_t*) blockReorderingArrayHelp, (uint32_t*) blockReorderingArray,  blockPartitionCounterUInt32, swapMoves);

		blockMergeArray[threadIdx.x].cost = 0xffffffff;
		for (uint32_t i = threadIdx.x; i < swapMoves; i += blockDim.x)	{
			struct MoveIndices *move = &blockReorderingArray[i];
			uint32_t threadBestCost = blockMergeArray[threadIdx.x].cost;
			uint32_t totalEval = cudaEvaluateOrder(cudaData, blockCurrentOrder, move->i, move->j, blockActivitiesDuration, blockResourceIndices, threadResourcesLoad,
					threadStartValues, threadRemainingResourcesCapacity, threadStartTimesById, cudaData.capacityResolutionAlgorithm);
			totalEval = (totalEval > 0x0000ffff ? 0xffff0000 : totalEval<<16);
			uint32_t tardinessPenalty = cudaComputeTardinessPenalty(cudaData.numberOfActivities, blockActivitiesDuration, blockBestCost-1, threadStartTimesById);
			tardinessPenalty = (tardinessPenalty > 0xfff ? 0xfff : tardinessPenalty);
			totalEval |= ((tardinessPenalty<<4) & 0x0000fff0);
			totalEval |= (hiprand(&threadRandState) & 0x0000000f);

			bool isPossibleMove = cudaIsPossibleMove(cudaData.numberOfActivities, move->i, move->j, blockTabuCache);
			if ((isPossibleMove && totalEval < threadBestCost) || (totalEval>>16) < blockBestCost)	{
				struct MoveInfo newBestThreadSolution = { move->i, move->j, totalEval };
				blockMergeArray[threadIdx.x] = newBestThreadSolution;
			}
		}
		
		if (threadIdx.x == 0)	{
			blockUniformProbability = hiprand_uniform(&randState);
		}
		__syncthreads();

		if (blockUniformProbability > 0.6f*((float) blockIterationsSinceImprovement)/((float) blockMaximalNumberOfIterationsSinceLoad))	{
			for (uint16_t k = blockDim.x/2; k > 0; k >>= 1)	{
				if (threadIdx.x < k)	{
					if (blockMergeArray[threadIdx.x].cost > blockMergeArray[threadIdx.x+k].cost)
						blockMergeArray[threadIdx.x] = blockMergeArray[threadIdx.x+k];
				}
				__syncthreads();
			}
		} else if (threadIdx.x == 0)	{
			blockMergeArray[0] = blockMergeArray[hiprand(&randState) % blockDim.x];
		}
		__syncthreads();

		if (threadIdx.x == 0)	{
			blockReadPossible = false;
			iterBestMove = blockMergeArray[0];
			iterBestMove.cost >>= 16;
			atomicAdd((unsigned long long*) cudaData.evaluatedSchedules, swapMoves);
			atomicInc(&cudaData.infoAboutSolutions[blockIndexOfSetSolution].iterationCounter, 0xffffffff);
			if (iterBestMove.cost < blockBestCost)	{
				blockWriteBestBlock = true;
				blockBestCost = iterBestMove.cost;
				blockIterationsSinceImprovement = 0;
			} else {
				++blockIterationsSinceImprovement;
			}
			++blockNumberOfIterationsSinceLoad;

			uint32_t readSlotCost = cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost;
			if (blockNumberOfIterationsSinceLoad >= blockMaximalNumberOfIterationsSinceLoad || readSlotCost != blockBestCost || *cudaData.bestSolutionCost == cudaData.criticalPathLength) {
				bool setOfSolutionsAccess = false;
				if (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					setOfSolutionsAccess = true;

				if (setOfSolutionsAccess)	{

					if (blockBestCost < cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost)	{
						blockWriteSetSolution = true;
						cudaData.infoAboutSolutions[blockIndexOfSetSolution].readCounter = 0;
						cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost = blockBestCost;
						if (blockBestCost < *cudaData.bestSolutionCost)	{
							*cudaData.bestSolutionCost = blockBestCost;
							*cudaData.indexToTheBestSolution = blockIndexOfSetSolution;
						}
					}	else	{
						atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
					}

					if (*cudaData.bestSolutionCost == cudaData.criticalPathLength)	{
						blockCriticalPathLengthAchieved = true;
					}

					if (readSlotCost < blockBestCost || blockNumberOfIterationsSinceLoad >= blockMaximalNumberOfIterationsSinceLoad)	{
						blockReadSetSolution = true;
					}
				}
			}
		}
		__syncthreads();
		
		if (blockMergeArray[0].cost == 0xffffffff)	{
			// Empty expanded neighborhood. Tabu list will be pruned.
			cudaClearTabuList(cudaData.numberOfActivities, blockTabuList, blockTabuCache, blockTabuListSize/3);
		} else if (threadIdx.x == 0)	{
			// Apply best move.
			uint16_t t = blockCurrentOrder[iterBestMove.i];
			blockCurrentOrder[iterBestMove.i] = blockCurrentOrder[iterBestMove.j];
			blockCurrentOrder[iterBestMove.j] = t;
			// Add move to tabu list.
			cudaAddTurnToTabuList(cudaData.numberOfActivities, iterBestMove.i, iterBestMove.j, blockTabuList, blockTabuCache, blockTabuIdx, blockTabuListSize);
		}

		if (blockWriteBestBlock == true)	{
			__syncthreads();
			for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
				blockBestSolution[i] = blockCurrentOrder[i];

			if (threadIdx.x == 0)
				blockWriteBestBlock = false;
		}
		__syncthreads();

		if (blockWriteSetSolution == true)	{
			for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
				cudaData.ordersOfSolutions[blockIndexOfSetSolution*cudaData.numberOfActivities+i] = blockBestSolution[i];
			for (uint16_t i = threadIdx.x; i < cudaData.maxTabuListSize; i += blockDim.x)
				cudaData.tabuListsOfSetOfSolutions[blockIndexOfSetSolution*cudaData.maxTabuListSize+i] = blockTabuList[i];

			__threadfence();
			__syncthreads();

			if (threadIdx.x == 0)	{
				blockWriteSetSolution = false; 
				atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
			}
		}

		if (blockReadSetSolution == true)	{
			if (threadIdx.x == 0)	{
				if (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) == DATA_AVAILABLE)
					blockReadPossible = true;
			}
			__syncthreads();
			if (blockReadPossible)	{
				if (threadIdx.x == 0)	{
					blockIndexOfSetSolution = (blockIndexOfSetSolution+1) % cudaData.totalSolutions;
				}
				__syncthreads();
				// Read a solution from a set to block memory.
				cudaReadExternalSolution(cudaData.numberOfActivities, blockTabuList, blockTabuCache, blockTabuListSize, blockCurrentOrder,
						cudaData.ordersOfSolutions+blockIndexOfSetSolution*cudaData.numberOfActivities, cudaData.tabuListsOfSetOfSolutions+blockIndexOfSetSolution*cudaData.maxTabuListSize);

				if (threadIdx.x == 0)	{
					blockBestCost = cudaData.infoAboutSolutions[blockIndexOfSetSolution].solutionCost;
					uint32_t readCounter = ++cudaData.infoAboutSolutions[blockIndexOfSetSolution].readCounter;
					blockNumberOfIterationsSinceLoad = blockIterationsSinceImprovement = 0;

					blockReadSetSolution = false;
					blockMaximalNumberOfIterationsSinceLoad = calculateTheNumberOfAssignedIterationsSinceLoad(cudaData, blockIndexOfSetSolution);
					atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);
					if (readCounter > cudaData.maximalValueOfReadCounter)
						cudaDiversificationOfSolution(cudaData, blockCurrentOrder, blockSuccessorsMatrix, cudaData.numberOfDiversificationSwaps, &randState);
				}
			}
		}

		if (threadIdx.x == 0)	{
			++iter;
		}
		__syncthreads();
	}

	// Write solution if is better than the best found.
	if (threadIdx.x == 0)	{
		while (atomicCAS(cudaData.lockSetOfSolutions, DATA_AVAILABLE, DATA_ACCESS) != DATA_AVAILABLE)
			;
	}
	__syncthreads();

	if (*cudaData.bestSolutionCost > blockBestCost)	{
		for (uint16_t i = threadIdx.x; i < cudaData.numberOfActivities; i += blockDim.x)
			cudaData.ordersOfSolutions[blockIndexOfSetSolution*cudaData.numberOfActivities+i] = blockBestSolution[i];
		for (uint16_t i = threadIdx.x; i < cudaData.maxTabuListSize; i += blockDim.x)
			cudaData.tabuListsOfSetOfSolutions[blockIndexOfSetSolution*cudaData.maxTabuListSize+i] = blockTabuList[i];
		if (threadIdx.x == 0)	{
			*cudaData.bestSolutionCost = blockBestCost;
			*cudaData.indexToTheBestSolution = blockIndexOfSetSolution;
		}
	}
	__threadfence();
	__syncthreads();

	if (threadIdx.x == 0)
		atomicExch(cudaData.lockSetOfSolutions, DATA_AVAILABLE);

	return;
}


/* START MAIN CUDA KERNEL */

void runCudaSolveRCPSP(int numberOfBlock, int numberOfThreadsPerBlock, int computeCapability, int dynSharedMemSize, const CudaData& cudaData)	{
	if (computeCapability < 300)	{
		if (dynSharedMemSize < 7950)	{
			// 16 kB shared memory + 48 kB cache L1.
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaSolveRCPSP), hipFuncCachePreferL1);
		} else {
			// 48 kB shared memory + 16 kB cache L1.
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaSolveRCPSP), hipFuncCachePreferShared);
		}
	} else {
		// 32 kB shared memory + 32 kB cache L1
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cudaSolveRCPSP), hipFuncCachePreferEqual);
	}
	// Launch the main GPU kernel.
	cudaSolveRCPSP<<<numberOfBlock,numberOfThreadsPerBlock,dynSharedMemSize>>>(cudaData);
	hipDeviceSynchronize();
}

